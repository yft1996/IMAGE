#include "hip/hip_runtime.h"
/*
 * @功能      image.hpp内TODO函数实现
 * @姓名      杨丰拓
 * @日期      2019-4-29
 * @时间      17:14
 * @邮箱
*/
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include "cuda_include/common.cuh"
#include "cuda_include/sharedmem.cuh"
#include <cstdio>
template <typename T>
void gpu_cpu2zero1(T *cpu,T *gpu,size_t bytes)
{
    memset(cpu, 0, bytes);
    hipMemset(gpu,0,bytes);
}


/******************************************************************************************/
///功能：填充图像
/*  函数名                           线程块大小       耗费时间
 *  kernel_fill_color	            702.651us	    [32,4,1]
 *  kernel_fill_color3	            705.469us	    [32,16,1]
 *  kernel_fill_color3_by_share	    400.097us	    [32,4,1]
 *  kernel_fill_color15_by_share	253.638us	    [32,4,1]**
 */
///核函数
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((wc-1+x)/x,(h-1+y)/y,1);
 * kernel_fill_color<T><<<grid,block>>>(d_out,d_color,wc,h,c);
 */
template <typename T>
__global__ void kernel_fill_color(T * image, T *color,int const wc,int const h,int const c)
{
    int x=threadIdx.x+blockIdx.x*blockDim.x;
    int y=threadIdx.y+blockIdx.y*blockDim.y;
    int idx=y*wc+x;
    //越界判断
    if(x<wc&&y<h)
    {
        int channels=idx%c;
        image[idx]=color[channels];
    }
}
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((wc-1+x*3)/(x*3),(h-1+y)/y,1);
 * kernel_fill_color3<T><<<grid,block>>>(d_out,d_color,wc,h,c);
 */
template <typename T>
__global__ void kernel_fill_color3(T * image, T *color,int const wc,int const h,int const c)
{
    int x=threadIdx.x+blockIdx.x*blockDim.x*3;
    int y=threadIdx.y+blockIdx.y*blockDim.y;
    int idx=y*wc+x;

    T local_color[4];
    for(int i=0;i<c;i++)
    {
        local_color[i]=color[i];
    }
    //越界判断
    if((x+blockDim.x*2)<wc&&y<h)
    {
        int channels=idx%c;
        image[idx]=local_color[channels];

        idx+=blockDim.x;
        channels=idx%c;
        image[idx]=local_color[channels];

        idx+=blockDim.x;
        channels=idx%c;
        image[idx]=local_color[channels];
    }
}
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((wc-1+x*3)/(x*3),(h-1+y)/y,1);
 * kernel_fill_color3_by_share<T><<<grid,block,colorbytes>>>(d_out,d_color,wc,h,c);
 */
template <typename T>
__global__ void kernel_fill_color3_by_share(T * image, T *color,int const wc,int const h,int const c)
{
    SharedMemory<T> smem;
    T* data = smem.getPointer();
    int x=threadIdx.x+blockIdx.x*blockDim.x*3;
    int y=threadIdx.y+blockIdx.y*blockDim.y;
    int idx=y*wc+x;
    int sidx=threadIdx.y*blockDim.x+threadIdx.x;
    if(sidx<c)data[sidx]=color[sidx];
    __syncthreads();
    //越界判断
    if((x+blockDim.x*2)<wc&&y<h)
    {
        int channels;
        for(int k=0;k<3;k++)
        {
            channels=idx%c;
            image[idx]=data[channels];
            idx+=blockDim.x;
        }
    }
}
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((wc-1+x*15)/(x*15),(h-1+y)/y,1);
 * kernel_fill_color15_by_share<T><<<grid,block,colorbytes>>>(d_out,d_color,wc,h,c);
 */
template <typename T>
__global__ void kernel_fill_color15_by_share(T * image, T *color,int const wc,int const h,int const c)
{
    SharedMemory<T> smem;
    T* data = smem.getPointer();
    int x=threadIdx.x+blockIdx.x*blockDim.x*15;
    int y=threadIdx.y+blockIdx.y*blockDim.y;
    int idx=y*wc+x;
    int sidx=threadIdx.y*blockDim.x+threadIdx.x;
    if(sidx<c)data[sidx]=color[sidx];
    __syncthreads();
    //越界判断

    if(x<wc&&y<h)
    {
        int channels;
        for(int k=0;k<15;k++)
        {
            channels=idx%c;
            image[idx]=data[channels];
            idx+=blockDim.x;
        }
    }
}

///功能:添加颜色通道
/*  函数名                           线程块大小       耗费时间
 *  kernel_add_channels	            1.131ms	        [32,4,1]
 *  kernel_add_channels_stride	    507.197us	    [32,4,1]
 *  kernel_add_channels_stride2	    422.649us	    [32,4,1]**
 */
///核函数
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((w*c_add-1+x)/(x),(h-1+y)/y,1);
 * kernel_add_channels<T><<<grid,block>>>(d_out,d_in,w,h,c,num_channels,value);
 */
template <typename T>
__global__ void kernel_add_Channel(T *dst,T *src, int const w,int const h,int const c,int const num_channels,T value)
{
    int x=threadIdx.x+blockIdx.x*blockDim.x;//x坐标索引
    int y=threadIdx.y+blockIdx.y*blockDim.y;//y坐标索引
    int c_add=c+num_channels;
    int idx=y*w*c_add+x;//输出索引
    if(x<w*c_add&&y<h)
    {
        int channels=idx%c_add;
        int pixels=idx/c_add;
        if (channels < c) dst[idx] = src[pixels * c + channels];
        else dst[idx] = value[channels - c];
    }
}
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((w-1+x)/(x),(h-1+y)/y,1);
 * kernel_add_channels_stride<T><<<grid,block>>>(d_out,d_in,w,h,c,num_channels,value);
 */
template <typename T>
__global__ void kernel_add_Channel_stride(T *dst,T *src, int const w,int const h,int const c,int const num_channels,T value)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;//x坐标索引
    int y = threadIdx.y + blockIdx.y * blockDim.y;//y坐标索引
    int c_add=c+num_channels;
    int idx_out = y * w * c_add + x * c_add;//输出索引
    int idx_in = y * w * c + x * c;//输入索引
    if (x < w  && y < h)
    {
        for (int i = 0; i <c ; ++i) dst[idx_out+i]=src[idx_in+i];
        for (int j = 0; j <num_channels ; ++j)  dst[idx_out+c+j]=value[j];
    }
}
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((w-1+x*2)/(x*2),(h-1+y)/y,1);
 * kernel_add_channels_stride2<T><<<grid,block>>>(d_out,d_in,w,h,c,num_channels,value);
 */
template <typename T>
__global__ void kernel_add_Channel_stride2(T *dst,T *src, int const w,int const h,int const c,int const num_channels,T value)
{
    int x=threadIdx.x+blockIdx.x*blockDim.x*2;//x坐标索引
    int y=threadIdx.y+blockIdx.y*blockDim.y;//y坐标索引
    int c_add=c+num_channels;
    int idx_out=y*w*c_add+x*c_add;//输出索引
    int idx_in=y*w*c+x*c;//输入索引
    if (x < w  && y < h)
    {
        for (int i = 0; i <c ; ++i)
        {
            dst[idx_out+i]=src[idx_in+i];
            dst[idx_out+blockDim.x*c_add+i]=src[idx_in+blockDim.x*c+i];
        }
        for (int j = 0; j <num_channels ; ++j) {
            dst[idx_out + c + j] = value;
            dst[idx_out + blockDim.x * c_add + c + j] = value;
        }
    }
}

///功能:添加颜色通道(多颜色数据)
/*  函数名                           线程块大小       耗费时间
 *  kernel_add_channels	            1.131ms	        [32,4,1]
 *  kernel_add_channels_stride	    507.197us	    [32,4,1]
 *  kernel_add_channels_stride2	    422.649us	    [32,4,1]**
 */
///核函数
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((w*c_add-1+x)/(x),(h-1+y)/y,1);
 * kernel_add_channels<T><<<grid,block>>>(d_out,d_in,w,h,c,num_channels,d_value,_front_back);
 */
template <typename T>
__global__ void kernel_add_channels(T *dst,T *src, int const w,int const h,int const c,int const num_channels,T * value,bool _front_back)
{
    int x=threadIdx.x+blockIdx.x*blockDim.x;//x坐标索引
    int y=threadIdx.y+blockIdx.y*blockDim.y;//y坐标索引
    int c_add=c+num_channels;
    int idx=y*w*c_add+x;//输出索引
    if(x<w*c_add&&y<h)
    {
        int channels=idx%c_add;
        int pixels=idx/c_add;
        if(_front_back)
        {
            if (channels < c) dst[idx] = src[pixels * c + channels];
            else dst[idx] = value[channels - c];
        }
        else
        {
            if (channels < num_channels) dst[idx] = value[channels];
            else dst[idx] = src[pixels * c + channels - num_channels];
        }
    }
}
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((w-1+x)/(x),(h-1+y)/y,1);
 * kernel_add_channels_stride<T><<<grid,block>>>(d_out,d_in,w,h,c,num_channels,d_value,_front_back);
 */
template <typename T>
__global__ void kernel_add_channels_stride(T *dst,T *src, int const w,int const h,int const c,int const num_channels,T * value,bool _front_back)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;//x坐标索引
    int y = threadIdx.y + blockIdx.y * blockDim.y;//y坐标索引
    int c_add=c+num_channels;
    int idx_out = y * w * c_add + x * c_add;//输出索引
    int idx_in = y * w * c + x * c;//输入索引
    if (x < w  && y < h)
    {
        if(_front_back)
        {
            for (int i = 0; i <c ; ++i) dst[idx_out+i]=src[idx_in+i];
            for (int j = 0; j <num_channels ; ++j)  dst[idx_out+c+j]=value[j];
        }
        else
        {
            for (int j = 0; j <num_channels ; ++j)  dst[idx_out+j]=value[j];
            for (int i = 0; i <c ; ++i) dst[idx_out+num_channels+i]=src[idx_in+i];
        }
    }
}
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((w-1+x*2)/(x*2),(h-1+y)/y,1);
 * kernel_add_channels_stride2<T><<<grid,block>>>(d_out,d_in,w,h,c,num_channels,d_value,_front_back);
 */
template <typename T>
__global__ void kernel_add_channels_stride2(T *dst,T *src, int const w,int const h,int const c,int const num_channels,T * value,bool _front_back)
{
    int x=threadIdx.x+blockIdx.x*blockDim.x*2;//x坐标索引
    int y=threadIdx.y+blockIdx.y*blockDim.y;//y坐标索引
    int c_add=c+num_channels;
    int idx_out=y*w*c_add+x*c_add;//输出索引
    int idx_in=y*w*c+x*c;//输入索引
    if (x < w  && y < h)
    {
        if(_front_back)
        {
            for (int i = 0; i <c ; ++i)
            {
                dst[idx_out+i]=src[idx_in+i];
                dst[idx_out+blockDim.x*c_add+i]=src[idx_in+blockDim.x*c+i];
            }
            for (int j = 0; j <num_channels ; ++j)
            {
                dst[idx_out+c+j]=value[j];
                dst[idx_out+blockDim.x*c_add+c+j]=value[j];
            }
        }
        else
        {
            for (int j = 0; j <num_channels ; ++j)
            {
                dst[idx_out+j]=value[j];
                dst[idx_out+blockDim.x*c_add+j]=value[j];
            }
            for (int i = 0; i <c ; ++i)
            {
                dst[idx_out+num_channels+i]=src[idx_in+i];
                dst[idx_out+blockDim.x*c_add+num_channels+i]=src[idx_in+blockDim.x*c+i];
            }
        }
    }
}

///功能:交换颜色通道
/*  函数名                           线程块大小       耗费时间
 * kernel_swap_channels	            283.847us	   [32,4,1]**
 * kernel_swap_channels2	        293.352us	   [32,4,1]
 */
///核函数
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((w-1+x)/(x),(h-1+y)/y,1);
 * kernel_swap_channels<T><<<grid,block>>>(d_in,w,h,c,swap_c1,swap_c2);
 */
template <typename T>
__global__ void kernel_swap_channels(T *src,int const w,int const h,int const c, int const swap_c1,int const swap_c2)
{
    int const x=threadIdx.x+blockDim.x*blockIdx.x;
    int const y=threadIdx.y+blockDim.y*blockIdx.y;
    int const idx=y*w+x;
    if(x<w&&y<h)
    {
        T a,b;
        a=src[idx*c+swap_c1];
        b=src[idx*c+swap_c2];
        src[idx*c+swap_c1]=b;
        src[idx*c+swap_c2]=a;
    }
}
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((w-1+x)/(x),(h-1+y*2)/(y*2),1);
 * kernel_swap_channels2<T><<<grid,block>>>(d_in,w,h,c,swap_c1,swap_c2);
 */
template <typename T>
__global__ void kernel_swap_channels2(T *src,int const w,int const h,int const c, int const swap_c1,int const swap_c2)
{
    int  x=threadIdx.x+blockIdx.x*blockDim.x;
    int  y=threadIdx.y+blockIdx.y*blockDim.y*2;
    for(int i=0;i<2;i++)
    {
        int idx=(y+blockDim.y*i)*w*c+x*c;
        if(x<w&&(y+blockDim.y*i)<h)
        {
            T a,b;
            a=src[idx+swap_c1];
            b=src[idx+swap_c2];
            src[idx+swap_c1]=b;
            src[idx+swap_c2]=a;
        }
    }
}

///功能:复制颜色通道
/*  函数名                           线程块大小       耗费时间
 * kernel_copy_channels	            286.692us	    [32,4,1]**
 */
///核函数
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((w-1+x)/(x),(h-1+y)/y,1);
 * kernel_copy_channels<T><<<grid,block>>>(d_in,w,h,c,copy_c,paste_c);
 */
template <typename  T>
__global__ void kernel_copy_channels(T *image,int const w,int const h,int const c,int const copy_c,int const paste_c)
{
    int x=blockDim.x*blockIdx.x+threadIdx.x;
    int y=blockDim.y*blockIdx.y+threadIdx.y;
    if(x<w&&y<h)
    {
        int idx=y*w*c+x*c;
        T value=image[idx+copy_c];
        image[idx+paste_c]=value;
    }
}

///功能:删除颜色通道
/*  函数名                           线程块大小       耗费时间
 * kernel_delete_channel	        468.206us	   [32,4,1]
 * kernel_delete_channel2	        322.506us	   [32,2,1]**
 * kernel_delete_channel3	        334.987us	   [32,2,1]
 */
///核函数
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((src_w*dst_c-1+x*5)/(x*5),(src_h-1+y)/y,1);
 * kernel_delete_channel<T><<<grid,block>>>(d_out,d_in,src_w,src_h,src_c,dst_c,del_c);
 */
template <typename T>
__global__ void kernel_delete_channel(T *dst,T *src,int const w,int const h,int const c,int const dst_c,int const del_c)
{
    int x=threadIdx.x+blockIdx.x*blockDim.x*5;
    int y=threadIdx.y+blockIdx.y*blockDim.y;
    for (int i = 0; i <5 ; ++i) {

        if(x<w*dst_c&&y<h)
        {
            int idx_out=y*w*dst_c+x;
            int channel=idx_out%dst_c;
            int pixel=idx_out/dst_c;
            int idx_in=pixel*c+channel;
            T value;
            if(channel>=del_c)idx_in+=1;
            value=src[idx_in];
            dst[idx_out]=value;
        }
        x+=blockDim.x;
    }
}
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((src_w-1+x*2)/(x*2),(src_h-1+y)/y,1);
 * kernel_delete_channel2<T><<<grid,block>>>(d_out,d_in,src_w,src_h,src_c,dst_c,del_c);
 */
template <typename T>
__global__ void kernel_delete_channel2(T *dst,T *src,int const w,int const h,int const c,int const dst_c,int const del_c)
{
    int x=threadIdx.x+blockIdx.x*blockDim.x*2;
    int y=threadIdx.y+blockIdx.y*blockDim.y;
    if(x<w&&y<h)
    {
        int pixel=y*w+x;
        int pixel1=y*w+x+blockDim.x;
        T value;
        int j=0;
        for (int i = 0; i <c ; ++i)
        {
            if(i!=del_c)
            {
                value=src[pixel*c+i];
                dst[pixel*dst_c+j]=value;
                value=src[pixel1*c+i];
                dst[pixel1*dst_c+j]=value;
                j++;
            }
        }
    }
}
/* 调用示例
 * dim3 block(x,y,1);
 * dim3 grid((src_w-1+x*3)/(x*3),(src_h-1+y)/y,1);
 * kernel_delete_channel3<T><<<grid,block>>>(d_out,d_in,src_w,src_h,src_c,dst_c,del_c);
 */
template <typename T>
__global__ void kernel_delete_channel3(T *dst,T *src,int const w,int const h,int const c,int const dst_c,int const del_c)
{
    int x=threadIdx.x+blockIdx.x*blockDim.x*3;
    int y=threadIdx.y+blockIdx.y*blockDim.y;
    if(x<w&&y<h)
    {
        int pixel=y*w+x;
        int pixel2=pixel+blockDim.x;
        int pixel3=pixel2+blockDim.x;
        T value;
        int j=0;
        for (int i = 0; i <c ; ++i)
        {
            if(i!=del_c)
            {
                value=src[pixel*c+i];
                dst[pixel*dst_c+j]=value;
                value=src[pixel2*c+i];
                dst[pixel2*dst_c+j]=value;
                value=src[pixel3*c+i];
                dst[pixel3*dst_c+j]=value;
                j++;
            }
        }
    }
}




/******************************************************************************************/
///调用核函数实现加速功能
/******************************************************************************************/

///填充颜色通道函数
template <typename T>
int fill_color_cu(T *image,T *color,int const w,int const h,int const c,int const color_size)
{
    //bool flag= false;
    if(c!=color_size)
    {
        std::cerr<<"颜色通道不匹配"<<std::endl;
        return 0;
    }
    int wc=w*c;
    //定义显存指针
    T *d_out=NULL;
    T *d_color=NULL;
    //计算显存所需字节数
    size_t const imagebytes=w*h*c*sizeof(T);
    int const colorbytes=color_size* sizeof(T);
    //分配显存
    hipMalloc((void**)&d_out,imagebytes);
    hipMalloc((void**)&d_color,colorbytes);
    //cpu2gpu
    hipMemcpy(d_color,color,colorbytes,hipMemcpyHostToDevice);

    //线程网格划分
    int x=32;
    int y=4;
    dim3 block(x,y,1);
    dim3 grid((wc-1+x*15)/(x*15),(h-1+y)/y,1);

    kernel_fill_color15_by_share<T><<<grid,block,colorbytes>>>(d_out,d_color,wc,h,c);

    //gpu2cpu
    hipMemcpy(image,d_out,imagebytes,hipMemcpyDeviceToHost);
    //compare1<T>(image,contrast,w*c,h,flag);
    //释放显存
    hipFree(d_out);
    hipFree(d_color);
    return 0;
}
///增加颜色通道函数(单通道多数据)
template <typename T>
int add_Channel_cu(T *dst_image,T * src_image,int const w,int const h, int const c, int const num_channels,T  value)
{
    if(num_channels<=0)
    {
        std::cerr<<"所添加的颜色通道个数小于1"<<std::endl;
        return 0;
    }
    int const wc =w*c;//输入图像实际宽度
    int const wc_add=w*(c+num_channels);//输出图像实际宽度
    //计算存储空间字节数
    size_t const bytes_src=wc*h* sizeof(T);
    size_t const bytes_dst=wc_add*h* sizeof(T);
    //声明显存指针
    T *d_in=NULL,*d_out=NULL;
    //定义显存指针
    hipMalloc((void**)&d_in,bytes_src);
    hipMalloc((void**)&d_out,bytes_dst);
    //cpu2gpu
    hipMemcpy(d_in,src_image,bytes_src,hipMemcpyHostToDevice);
    //int c_add=c+num_channels;
    //网格划分
    int x=32;
    int y=4;
    dim3 block(x,y,1);
    dim3 grid((w-1+x*2)/(x*2),(h-1+y)/y,1);
    //核函数
    kernel_add_Channel_stride2<T><<<grid,block>>>(d_out,d_in,w,h,c,num_channels,value);
    //gpu2cpu
    hipMemcpy(dst_image,d_out,bytes_dst,hipMemcpyDeviceToHost);
    //compare1(dst_image,contrast,w*c,h, false);
    ///释放显存指针
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
///增加颜色通道函数(多通道多数据)
template <typename T>
int add_channels_cu(T *dst_image,T * src_image,int const w,int const h, int const c, int const num_channels,T * value,bool _front_back=true)
{
    if(num_channels<=0)
    {
        std::cerr<<"所添加的颜色通道个数小于1"<<std::endl;
        return 0;
    }
    int const wc =w*c;//输入图像实际宽度
    int const wc_add=w*(c+num_channels);//输出图像实际宽度
    //计算存储空间字节数
    size_t const bytes_value=num_channels* sizeof(T);
    size_t const bytes_src=wc*h* sizeof(T);
    size_t const bytes_dst=wc_add*h* sizeof(T);
    //声明显存指针
    T *d_in=NULL,*d_out=NULL,*d_value=NULL;
    //定义显存指针
    hipMalloc((void**)&d_value,bytes_value);
    hipMalloc((void**)&d_in,bytes_src);
    hipMalloc((void**)&d_out,bytes_dst);
    //cpu2gpu
    hipMemcpy(d_value,value,bytes_value,hipMemcpyHostToDevice);
    hipMemcpy(d_in,src_image,bytes_src,hipMemcpyHostToDevice);
    //int c_add=c+num_channels;
    //网格划分
    int x=32;
    int y=4;
    dim3 block(x,y,1);
    dim3 grid((w-1+x*2)/(x*2),(h-1+y)/y,1);
    //核函数
    kernel_add_channels_stride2<T><<<grid,block>>>(d_out,d_in,w,h,c,num_channels,d_value,_front_back);
    //gpu2cpu
    hipMemcpy(dst_image,d_out,bytes_dst,hipMemcpyDeviceToHost);
    //compare1(dst_image,contrast,w*c,h, false);
    ///释放显存指针
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_value);
    return 0;
}
///交换颜色通道函数
template <typename T>
int swap_channels_by_cu(T *src,int const w,int const h,int c,int const swap_c1,int swap_c2)
{
    if(swap_c1==swap_c2)return 0;
    if(swap_c1<0||swap_c1>=c||swap_c2<0||swap_c2>=c)
    {
        std::cerr<<"swap_channels_by_cuda函数所要交换的颜色通道不合适!!"<<std::endl;
        return 1;
    }
    //计算字节数
    size_t const bytes=w*h*c* sizeof(T);
    //声明显存指针
    T *d_in=NULL;
    //定义显存指针
    hipMalloc((void**)&d_in,bytes);
    //cpu2gpu
    hipMemcpy(d_in,src,bytes,hipMemcpyHostToDevice);
    //网格划分
    int x=32;
    int y=4;
    dim3 block(x,y,1);
    dim3 grid((w-1+x)/(x),(h-1+y)/y,1);
    //核函数
    kernel_swap_channels<T><<<grid,block>>>(d_in,w,h,c,swap_c1,swap_c2);
    //gpu2cpu
    hipMemcpy(src,d_in,bytes,hipMemcpyDeviceToHost);
    //释放显存指针
    hipFree(d_in);
    return 0;
}
///复制颜色通道
template <typename  T>
int copy_channels_by_cu(T *image,int const w,int const h,int const c,int const copy_c,int const paste_c)
{
    if(copy_c>=c||paste_c>=c)
    {
        std::cerr<<"输入通道数超过图像的最大通道数"<<std::endl;
        return 1;
    }
    if(copy_c==paste_c)return 0;
    if(paste_c<0)
    {
        //TODO:向后添加一个全为零的颜色通道
    }
    //计算字节数
    size_t const bytes=w*h*c* sizeof(T);
    //声明显存指针
    T *d_in=NULL;
    //定义显存指针
    hipMalloc(&d_in,bytes);
    //cpu2gpu
    hipMemcpy(d_in,image,bytes,hipMemcpyHostToDevice);
    //网格划分
    int x=32;
    int y=4;
    dim3 block(x,y,1);
    dim3 grid((w-1+x)/(x),(h-1+y)/y,1);
    //核函数
    kernel_copy_channels<T><<<grid,block>>>(d_in,w,h,c,copy_c,paste_c);
    //gpu2cpu
    hipMemcpy(image,d_in,bytes,hipMemcpyDeviceToHost);
    //释放显存指针
    hipFree(d_in);
    return 0;
}
///删除颜色通道
template <typename T>
int delete_channel_by_cu(T *dstImage,T *srcImage,int const src_w,int const src_h,int const src_c,int const del_c)
{
    if(del_c<0||del_c>=src_c)return 0;
    int const dst_c=src_c-1;//输出通道数
    //计算所需存储的字节数
    size_t const bytes_in=src_w*src_h*src_c* sizeof(T);
    size_t const bytes_out=src_w*src_h*dst_c* sizeof(T);
    //声明显存指针
    T *d_in=NULL;
    T *d_out=NULL;
    //定义显存指针
    hipMalloc(&d_in,bytes_in);
    hipMalloc(&d_out,bytes_out);
    //cpu2gpu
    hipMemcpy(d_in,srcImage,bytes_in,hipMemcpyHostToDevice);
    //网格划分
    int x=32;
    int y=2;
    dim3 block(x,y,1);
    dim3 grid((src_w-1+x*2)/(x*2),(src_h-1+y)/y,1);
    //核函数
    kernel_delete_channel2<T><<<grid,block>>>(d_out,d_in,src_w,src_h,src_c,dst_c,del_c);
    //gpu2cpu
    hipMemcpy(dstImage,d_out,bytes_out,hipMemcpyDeviceToHost);
    //释放显存指针
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}




/******************************************************************************************/
///调用函数模板化
/******************************************************************************************/
///填充颜色通道函数
template <typename T>
int fill_color_by_cuda(T *image,T *color,int const w,int const h,int const c,int const color_size,T *contrast)
{
    fill_color_cu<T>(image,color,w,h,c, color_size);
    return 0;
}
template <>
int fill_color_by_cuda(char *image,char *color,int const w,int const h,int const c,int const color_size,char *contrast)
{
    fill_color_cu<char>(image,color,w,h,c, color_size);
    //compare1<char>(image,contrast,w*c,h, false);
    return 0;
}
template <>
int fill_color_by_cuda(float  *image,float *color,int const w,int const h,int const c,int const color_size,float *contrast)
{
    fill_color_cu<float>(image,color,w,h,c, color_size);
    //compare1<float>(image,contrast,w*c,h, true);
    return 0;
}

///增加颜色通道函数(后)
template <typename T>
int add_channels_by_cuda(T *dst_image,T  * src_image,int const w,int const h, int const c, int const num_channels,T  value,T *contrast)
{
    add_Channel_cu(dst_image,src_image, w, h, c,num_channels,value);
    return  0;
}
template <>
int add_channels_by_cuda(char *dst_image,char  * src_image,int const w,int const h, int const c, int const num_channels,char  value,char *contrast)
{
    add_Channel_cu<char>(dst_image,src_image, w, h, c,num_channels,value);
    compare1<char>(dst_image,contrast,w*c,h, false);
    return  0;
}
template <>
int add_channels_by_cuda(float *dst_image,float  * src_image,int const w,int const h, int const c, int const num_channels,float  value,float *contrast)
{
    add_Channel_cu<float>(dst_image,src_image, w, h, c,num_channels,value);
    //compare1<float>(dst_image,contrast,w*c,h, true);
    return  0;
}

///增加颜色通道函数(前/后)
template <typename T>
int add_channels_front_by_cuda(T *dst_image,T  * src_image,int const w,int const h, int const c, vector<T> value,bool _front_back,T *contrast)
{
    add_channels_cu(dst_image,src_image, w, h, c,(int)value.size(),&value.at(0),_front_back);
    //compare1(dst_image,contrast,w*c,h, false);
    return 0;
}
template <>
int add_channels_front_by_cuda(char *dst_image,char  * src_image,int const w,int const h, int const c, vector<char> value,bool _front_back,char *contrast)
{
    add_channels_cu<char>(dst_image,src_image, w, h, c,(int)value.size(),&value.at(0),_front_back);
    compare1<char>(dst_image,contrast,w*c,h, false);
    return 0;
}
template <>
int add_channels_front_by_cuda(float *dst_image,float  * src_image,int const w,int const h, int const c, vector<float> value,bool _front_back,float *contrast)
{
    add_channels_cu<float>(dst_image,src_image, w, h, c,(int)value.size(),&value.at(0),_front_back);
    compare1<float>(dst_image,contrast,w*c,h, false);
    return 0;
}

///交换颜色通道
template <typename T>
int swap_channels_by_cuda(T *src,int const w,int const h,int c,int const swap_c1,int swap_c2,T *contrast)
{
    swap_channels_by_cu(src,w,h,c,swap_c1,swap_c2);
    return 0;
}
template <>
int swap_channels_by_cuda(char *src,int const w,int const h,int c,int const swap_c1,int swap_c2,char *contrast)
{
    swap_channels_by_cu<char>(src,w,h,c,swap_c1,swap_c2);
    //compare1<char>(src,contrast,w*c,h, false);
    return 0;
}
template <>
int swap_channels_by_cuda(float *src,int const w,int const h,int c,int const swap_c1,int swap_c2,float *contrast)
{
    swap_channels_by_cu<float>(src,w,h,c,swap_c1,swap_c2);
    //compare1<float>(src,contrast,w*c,h, true);
    return 0;
}

///复制颜色通道
template <typename T>
int copy_channels_by_cuda(T *image,int const w,int const h,int const c,int const copy_c,int const paste_c,T *contrast)
{
    return 0;
}
template <>
int copy_channels_by_cuda(char *image,int const w,int const h,int const c,int const copy_c,int const paste_c,char *contrast)
{
    copy_channels_by_cu<char>(image,w,h,c,copy_c,paste_c);
    compare1<char>(image,contrast,w*c,h, false);
    return 0;
}
template <>
int copy_channels_by_cuda(float *image,int const w,int const h,int const c,int const copy_c,int const paste_c,float *contrast)
{
    copy_channels_by_cu<float>(image,w,h,c,copy_c,paste_c);
    compare1<float>(image,contrast,w*c,h, true);
    return 0;
}

///删除颜色通道
template <typename T>
int delete_channel_by_cuda(T *dstImage,T *srcImage,int const src_w,int const src_h,int const src_c,int const del_c,T *contrast)
{
    return 0;
}
template <>
int delete_channel_by_cuda(char *dstImage,char *srcImage,int const src_w,int const src_h,int const src_c,int const del_c,char *contrast)
{
    delete_channel_by_cu<char>(dstImage,srcImage,src_w,src_h,src_c,del_c);
    compare1<char>(dstImage,contrast,src_w*(src_c-1),src_h,false);
    return 0;
}
template <>
int delete_channel_by_cuda(float *dstImage,float *srcImage,int const src_w,int const src_h,int const src_c,int const del_c,float *contrast)
{
    delete_channel_by_cu<float>(dstImage,srcImage,src_w,src_h,src_c,del_c);
    compare1<float>(dstImage,contrast,src_w*(src_c-1),src_h,true);
    return 0;
}