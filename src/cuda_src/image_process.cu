/**
 * @desc    图像处理函数加速
 * @author  杨丰拓
 * @date    2019-04-4
 * @email   yangfengtuo@163.com
*/
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

__global__ void warmup(void)
{}
/*
__global__ void kernel_desaturate_alpha(float *out,float const *in, const int size,const int type)
{
    extern __shared__   float s[];
    int in_idx = threadIdx.x  + blockIdx.x * blockDim.x * 8 ;
    int out_idx = threadIdx.x+ blockIdx.x * blockDim.x * 4 ;
    int tid=threadIdx.x;
    int stride=tid*4;
    int stride1=stride+blockDim.x*4;
    if (in_idx< size * 4)
    {
        s[tid]=in[in_idx];
        s[tid+blockDim.x]=in[in_idx+blockDim.x];
        s[tid+blockDim.x*2]=in[in_idx+blockDim.x*2];
        s[tid+blockDim.x*3]=in[in_idx+blockDim.x*3];
        s[tid+blockDim.x*4]=in[in_idx+blockDim.x*4];
        s[tid+blockDim.x*5]=in[in_idx+blockDim.x*5];
        s[tid+blockDim.x*6]=in[in_idx+blockDim.x*6];
        s[tid+blockDim.x*7]=in[in_idx+blockDim.x*7];
    }
    __syncthreads();

    if(type==0)
    {
        out[out_idx]=max(s[stride+0],max(s[stride+1],s[stride+2]));
        out[out_idx+blockDim.x*2]=max(s[stride1+0],max(s[stride1+1],s[stride1+2]));
    }
    if(type==1)
    {
        float const max_v = max(s[stride+0],max(s[stride+1],s[stride+2]));
        float const min_v = min(s[stride+0],min(s[stride+1],s[stride+2]));
        out[out_idx]=0.5f*(max_v+min_v);
        float const max_s = max(s[stride1+0],max(s[stride1+1],s[stride1+2]));
        float const min_s = min(s[stride1+0],min(s[stride1+1],s[stride1+2]));
        out[out_idx+blockDim.x*2]=0.5f*(max_s+min_s);
    }
    if(type==2)
    {
        out[out_idx]=0.21f * s[stride+0] + 0.72f * s[stride+1] + 0.07f * s[stride+2];
        out[out_idx+blockDim.x*2]=0.21f * s[stride1+0] + 0.72f * s[stride1+1] + 0.07f * s[stride1+2];
    }
    if(type==3)
    {
        out[out_idx]=0.30f * s[stride+0] + 0.59f * s[stride+1] + 0.11f * s[stride+2];
        out[out_idx+blockDim.x*2]=0.30f * s[stride1+0] + 0.59f * s[stride1+1] + 0.11f * s[stride1+2];
    }
    if(type==4)
    {
        out[out_idx]=((float)(s[stride+0] + s[stride+1] + s[stride+2])) / 3.0f;
        out[out_idx+blockDim.x*2]=((float)(s[stride1+0] + s[stride1+1] + s[stride1+2])) / 3.0f;
    }
    out[out_idx+tid+1]=s[stride+3];
    out[out_idx+blockDim.x*2+tid+1]=s[stride1+3];
}
__global__ void kernel_desaturate(float *out,float const *in, const int size,const int type)
{
    extern __shared__   float s[];
    int in_idx = threadIdx.x  + blockIdx.x * blockDim.x * 6 ;
    int out_idx = threadIdx.x+ blockIdx.x * blockDim.x * 2 ;
    int tid=threadIdx.x;
    int stride=tid*3;
    int stride1=stride+blockDim.x*3;

    if (in_idx< size * 3)
    {
        s[tid]=in[in_idx];
        s[tid+blockDim.x]=in[in_idx+blockDim.x];
        s[tid+blockDim.x*2]=in[in_idx+blockDim.x*2];
        s[tid+blockDim.x*3]=in[in_idx+blockDim.x*3];
        s[tid+blockDim.x*4]=in[in_idx+blockDim.x*4];
        s[tid+blockDim.x*5]=in[in_idx+blockDim.x*5];
    }
    __syncthreads();
    if(type==0)
    {
        out[out_idx]=max(s[stride+0],max(s[stride+1],s[stride+2]));
        out[out_idx+blockDim.x]=max(s[stride1+0],max(s[stride1+1],s[stride1+2]));
    }
    if(type==1)
    {
        float const max_v = max(s[stride+0],max(s[stride+1],s[stride+2]));
        float const min_v = min(s[stride+0],min(s[stride+1],s[stride+2]));
        out[out_idx]=0.5f*(max_v+min_v);
        float const max_s = max(s[stride1+0],max(s[stride1+1],s[stride1+2]));
        float const min_s = min(s[stride1+0],min(s[stride1+1],s[stride1+2]));
        out[out_idx+blockDim.x]=0.5f*(max_s+min_s);
    }
    if(type==2)
    {
        out[out_idx]=0.21f * s[stride+0] + 0.72f * s[stride+1] + 0.07f * s[stride+2];
        out[out_idx+blockDim.x]=0.21f * s[stride1+0] + 0.72f * s[stride1+1] + 0.07f * s[stride1+2];
    }
    if(type==3)
    {
        out[out_idx]=0.30f * s[stride+0] + 0.59f * s[stride+1] + 0.11f * s[stride+2];
        out[out_idx+blockDim.x]=0.30f * s[stride1+0] + 0.59f * s[stride1+1] + 0.11f * s[stride1+2];
    }
    if(type==4)
    {
        out[out_idx]=((float)(s[stride+0] + s[stride+1] + s[stride+2])) / 3.0f;
        out[out_idx+blockDim.x]=((float)(s[stride1+0] + s[stride1+1] + s[stride1+2])) / 3.0f;
    }

}


void desaturate_by_cuda(float  * const out_image,float const *in_image,const int pixel_amount, const int type,const bool alpha)
{
    float *d_in=NULL;
    float *d_out=NULL;

    int bytes_in=pixel_amount*(3+alpha)*sizeof(float);
    int bytes_out=pixel_amount*(1+alpha)* sizeof(float);
    const int  blocksize=256;
    dim3 block(blocksize,1,1);
    dim3 grid((pixel_amount-1+blocksize*2)/(blocksize*2),1,1);
    cudaMalloc(&d_in,bytes_in);
    cudaMalloc(&d_out,bytes_out);
    cudaMemcpy(d_in,in_image,bytes_in,cudaMemcpyHostToDevice);
    if(alpha)
    {
        kernel_desaturate_alpha<<<grid,block,blocksize*4* sizeof(float)>>>(d_out,d_in,pixel_amount,type);
    }
    else
    {
        kernel_desaturate<<<grid,block,blocksize*6* sizeof(float)>>>(d_out,d_in,pixel_amount,type);
    }
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);

    cudaFree(d_in);
    cudaFree(d_out);
}
*/


/******************************************************************************************/
///功能：图片放大两倍
/*  函数名                         线程块大小       耗费时间
 *  kernel_doublesize             [32,16,1]      3.719ms(最快)
 *  kernel_doublesize_dim3        [32,8,3]       5.76ms
 *  kernel_doublesizebyshare      [32,32,1]      4.498ms
 */
/******************************************************************************************/
__global__ void kernel_doublesize(float *out,float *in,int const image_x,int const image_y,int const iw,int const ic)
{
    int out_x = threadIdx.x + blockIdx.x * blockDim.x * ic*2;
    int out_y = threadIdx.y + blockIdx.y * blockDim.y;
    for (int c = 0; c <ic*2 ; ++c)
    {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<image_y&&fact_x<image_x*ic)
        {
            int idx=fact_x+out_y*image_x*ic;
            bool nexty=(out_y+1)<image_y;
            bool nextx=(fact_x+ic)<(image_x*ic);
            int yoff[2]={ic*iw*(out_y>>1),
                         ic*iw*((out_y+nexty)>>1)};
            int xoff[2]={((fact_x/ic)>>1)*ic+fact_x%ic,
                         (((fact_x/ic)+nextx)>>1)*ic+fact_x%ic};
            int index[4]={yoff[0]+xoff[0],
                          yoff[0]+xoff[1],
                          yoff[1]+xoff[0],
                          yoff[1]+xoff[1]};
            out[idx]=0.25f*(in[index[0]]+in[index[1]]+in[index[2]]+in[index[3]]);
        }
    }
}
__global__ void kernel_doublesize_dim3(float *out,float *in,int const image_x,int const image_y,int const iw)
{
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*2;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int out_z=threadIdx.z;

    if(out_x<image_x&&out_y<image_y)
    {
        int idx=out_y*image_x*blockDim.z+out_x*blockDim.z+out_z;

        const bool nexty=(out_y+1)<image_y;
        const bool nextx=(out_x+1)<image_x;
        int yoff[2]={blockDim.z*iw*(out_y>>1),blockDim.z*iw*((out_y+nexty)>>1)};
        int xoff[2]={blockDim.z*(out_x>>1),blockDim.z*((out_x+nextx)>>1)};
        int index[4]={yoff[0]+xoff[0]+out_z,
                      yoff[0]+xoff[1]+out_z,
                      yoff[1]+xoff[0]+out_z,
                      yoff[1]+xoff[1]+out_z};
        out[idx]=0.25f*(in[index[0]]+in[index[1]]+in[index[2]]+in[index[3]]);

        int idx_2=out_y*image_x*blockDim.z+(out_x+blockDim.x)*blockDim.z+out_z;
        const bool nextx_2=(out_x+blockDim.x+1)<image_x;
        int xoff_2[2]={blockDim.z*((out_x+blockDim.x)>>1),blockDim.z*((out_x+blockDim.x+nextx_2)>>1)};
        int index_2[4]={yoff[0]+xoff_2[0]+out_z,
                      yoff[0]+xoff_2[1]+out_z,
                      yoff[1]+xoff_2[0]+out_z,
                      yoff[1]+xoff_2[1]+out_z};
        out[idx_2]=0.25f*(in[index_2[0]]+in[index_2[1]]+in[index_2[2]]+in[index_2[3]]);


    }

}

__global__ void kernel_doublesizebyshare(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    extern __shared__ float  data[];
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic*3;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;

    int share_x=(blockDim.x>>1)+1;//共享内存块x维（需乘ic）
    int share_y=(blockDim.y>>1)+1;//共享内存块y维
    int share_fact_x=share_x*3*ic;
    int share_idx_x;
    int share_idx_y= threadIdx.y;//共享内存块内y维索引
    int in_x0 = ((blockIdx.x * blockDim.x*3) >> 1) * ic;
    int in_y0 = (blockIdx.y * blockDim.y) >> 1;
    int x,y,c,fact_x;

    for ( c = 0; c <ic*3 ; ++c)
    {
        share_idx_x = threadIdx.x + blockDim.x * c;//共享内存块内x索引
        if (share_idx_x < share_fact_x && share_idx_y < share_y)
        {
            x = min(in_x0 + share_idx_x, iw * ic - ic + share_idx_x % ic);
            y = min(in_y0 + share_idx_y, ih - 1);
            data[share_idx_y * share_fact_x + share_idx_x] = in[y * iw * ic + x];
        }

    }
    __syncthreads();
    for ( c = 0; c <ic*3 ; ++c)
    {
        fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic)
        {
            share_idx_x = threadIdx.x + blockDim.x * c;
            int yoff[2]={(share_idx_y>>1)*share_fact_x,((share_idx_y+1)>>1)*share_fact_x};
            int xoff[2]={(share_idx_x/ic>>1)*ic+share_idx_x%ic,
                         ((share_idx_x/ic+1)>>1)*ic+share_idx_x%ic};
            int out_idx=out_y*ow*ic+fact_x;
            int index[4]={yoff[0]+xoff[0],
                          yoff[0]+xoff[1],
                          yoff[1]+xoff[0],
                          yoff[1]+xoff[1]};
            out[out_idx]=0.25f*(data[index[0]]+data[index[1]]+data[index[2]]+data[index[3]]);
        }
    }
    /*
    extern __shared__ float  data[];
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;

    int share_x=(blockDim.x>>1)+1;//共享内存块x维（需乘ic）
    int share_y=(blockDim.y>>1)+1;//共享内存块y维
    int share_idx_x;
    int share_idx_y;

    for (int c = 0; c <ic ; ++c) {
        share_idx_x = threadIdx.x + blockDim.x * c;//共享内存块内x索引
        share_idx_y = threadIdx.y;//共享内存块内y维索引

        int in_x0 = ((blockIdx.x * blockDim.x) >> 1) * ic;
        int in_y0 = (blockIdx.y * blockDim.y) >> 1;

        if (share_idx_x < (share_x * ic) && share_idx_y < share_y)
        {
            int x = min(in_x0 + share_idx_x, iw * ic - ic + share_idx_x % ic);
            int y = min(in_y0 + share_idx_y, ih - 1);
            data[share_idx_y * share_x * ic + share_idx_x] = in[y * iw * ic + x];
        }

    }
    __syncthreads();
    for (int c = 0; c <ic ; ++c) {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic)
        {
            share_idx_x = threadIdx.x + blockDim.x * c;
            share_idx_y = threadIdx.y;

            int yoff[2]={(share_idx_y>>1)*share_x*ic,((share_idx_y+1)>>1)*share_x*ic};
            int xoff[2]={(share_idx_x/ic>>1)*ic+share_idx_x%ic,
                         ((share_idx_x/ic+1)>>1)*ic+share_idx_x%ic};
            float val[4]={data[yoff[0]+xoff[0]],
                          data[yoff[0]+xoff[1]],
                          data[yoff[1]+xoff[0]],
                          data[yoff[1]+xoff[1]]};
            int out_idx=out_y*ow*ic+fact_x;
            out[out_idx]=0.25f*(val[0]+val[1]+val[2]+val[3]);
        }
    }*/

}


/******************************************************************************************/
///功能：图片缩小两倍
/*  函数名                         线程块大小       耗费时间
 *  kernel_halfsize1              [32,4,1]      639.142us
 *  kernel_halfsizebyshare1       [32,4,1]      654.107us
 *  kernel_halfsize               [32,8,1]      639.56us
 *  kernel_halfsizebyshare        [32,4,1]      687.768us
 */
/******************************************************************************************/
__global__ void kernel_halfsizebyshare1(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    extern __shared__ float data[];
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic*2;//输出的x维起始索引
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;//输出的y位索引
    int stride=iw*ic;//输入图像的行索引的最大值

    int in_x0=blockIdx.x*blockDim.x*2*ic*2;//输入图像x维的第一起始点
    int in_y0=blockIdx.y*blockDim.y*2;//输入图像y维的第一起始点
    int in_x1=in_x0+blockDim.x*ic*2;//输入图像x维的第二起始点
    int in_y1=in_y0+blockDim.y;//输入图像y维的第二起始点

    int share_x=blockDim.x*4;//共享块内x维最大像素点个数
    for (int c = 0; c < ic*2; ++c)
    {
        int fact_x_s=threadIdx.x+blockDim.x*c;//共享内存内第一个x的索引
        int x_s=fact_x_s+blockDim.x*ic*2;//共享内存内第二个x的索引
        int y_s0=threadIdx.y*share_x*ic;//共享内存内第一个y的索引
        int y_s1=y_s0+blockDim.y*share_x*ic;//共享内存内第二个y的索引
        int fact_iw=fact_x_s%ic+stride-ic;

        int x0=min(in_x0+fact_x_s,fact_iw);
        int x1=min(in_x1+fact_x_s,fact_iw);
        int y0=min(in_y0+threadIdx.y,ih-1)*stride;
        int y1=min(in_y1+threadIdx.y,ih-1)*stride;
        data[y_s0+fact_x_s]=in[y0+x0];
        data[y_s0+x_s]=in[y0+x1];
        data[y_s1+fact_x_s]=in[y1+x0];
        data[y_s1+x_s]=in[y1+x1];
    }
    __syncthreads();
    for (int c = 0; c <ic*2 ; ++c) {
        int fact_x=out_x+blockDim.x*c;

        if(out_y<oh&&fact_x<ow*ic)
        {
            int fact_x_s=threadIdx.x+blockDim.x*c;
            int srow1=threadIdx.y*2*share_x*ic;
            int srow2=srow1+share_x*ic;
            int scol1=(fact_x_s / ic) * 2 * ic + fact_x_s % ic;
            int scol2=scol1 + ic;
            int index[4] = {srow1 + scol1,
                            srow1 + scol2,
                            srow2 + scol1,
                            srow2 + scol2};
            int out_idx = out_y * ow*ic + fact_x;
            out[out_idx] = 0.25f * (data[index[0]] + data[index[1]] + data[index[2]] + data[index[3]]);
        }
    }
}
__global__ void kernel_halfsize(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int stride=iw*ic;

    for(int c=0;c<ic;c++)
    {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic) {
            int irow1 = out_y * 2 * stride;
            int irow2 = irow1 + stride * (out_y * 2 + 1 < ih);
            int icol1 = (fact_x / ic) * 2 * ic + fact_x % ic;
            int icol2 = min((icol1 + ic), (iw * ic - ic + fact_x % ic));
            int index[4] = {irow1 + icol1,
                            irow1 + icol2,
                            irow2 + icol1,
                            irow2 + icol2};
            int out_idx = out_y * ow*ic + fact_x;
            out[out_idx] = 0.25f * (in[index[0]] + in[index[1]] + in[index[2]] + in[index[3]]);
        }

    }
}
__global__ void kernel_halfsize1(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    //若需要展开ic*3重循环只需修改out_x=threadIdx.x+blockIdx.x*blockDim.x*ic*3;以及for(int c=0;c<ic*3;c++)即可，同时应修改网格大小
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic*2;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int stride=iw*ic;

    for(int c=0;c<ic*2;c++)
    {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic) {
            int irow1 = out_y * 2 * stride;
            int irow2 = irow1 + stride * (out_y * 2 + 1 < ih);
            int icol1 = (fact_x / ic) * 2 * ic + fact_x % ic;
            int icol2 = min((icol1 + ic), (iw * ic - ic + fact_x % ic));
            int index[4] = {irow1 + icol1,
                            irow1 + icol2,
                            irow2 + icol1,
                            irow2 + icol2};
            int out_idx = out_y * ow*ic + fact_x;
            out[out_idx] = 0.25f * (in[index[0]] + in[index[1]] + in[index[2]] + in[index[3]]);
        }

    }
}
__global__ void kernel_halfsizebyshare(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    extern __shared__ float data[];
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic;//输出的x维起始索引
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;//输出的y位索引
    int stride=iw*ic;//输入图像的行索引的最大值

    int in_x0=blockIdx.x*blockDim.x*2*ic;//输入图像x维的起始点
    int in_y0=blockIdx.y*blockDim.y*2;//输入图像y维的起始点
    int in_x1=in_x0+blockDim.x*ic;
    int in_y1=in_y0+blockDim.y;

    int share_x=blockDim.x*2;//共享块内x维最大像素点个数
    for (int c = 0; c < ic; ++c)
    {
        int fact_x_s=threadIdx.x+blockDim.x*c;
        int x_s=fact_x_s+blockDim.x*ic;
        int y_s0=threadIdx.y*share_x*ic;
        int y_s1=y_s0+blockDim.y*share_x*ic;
        int fact_iw=fact_x_s%ic+stride-ic;
        int x0=min(in_x0+fact_x_s,fact_iw);
        int x1=min(in_x1+fact_x_s,fact_iw);
        int y0=min(in_y0+threadIdx.y,ih-1)*stride;
        int y1=min(in_y1+threadIdx.y,ih-1)*stride;
        data[y_s0+fact_x_s]=in[y0+x0];
        data[y_s0+x_s]=in[y0+x1];
        data[y_s1+fact_x_s]=in[y1+x0];
        data[y_s1+x_s]=in[y1+x1];
    }
    __syncthreads();
    for (int c = 0; c <ic ; ++c)
    {
        int fact_x=out_x+blockDim.x*c;

        if(out_y<oh&&fact_x<ow*ic)
        {
            int fact_x_s=threadIdx.x+blockDim.x*c;
            int srow1=threadIdx.y*2*share_x*ic;
            int srow2=srow1+share_x*ic;
            int scol1=(fact_x_s / ic) * 2 * ic + fact_x_s % ic;
            int scol2=scol1 + ic;
            int index[4] = {srow1 + scol1,
                            srow1 + scol2,
                            srow2 + scol1,
                            srow2 + scol2};
            int out_idx = out_y * ow*ic + fact_x;
            out[out_idx] = 0.25f * (data[index[0]] + data[index[1]] + data[index[2]] + data[index[3]]);
        }
    }
}


/******************************************************************************************/
///调用核函数实现加速功能
/******************************************************************************************/
void warm(void)
{
    warmup<<<1,1>>>();
}

void double_size_by_cuda(float * const out_image,float const  * const in_image, int const weight,int const height,int const channels)
{
    int const ow=weight<<1;
    int const oh=height<<1;
    int const size_in=weight*height;
    int const size_out=ow*oh;
    int const bytes_in=size_in*channels* sizeof(float);
    int const bytes_out=size_out*channels* sizeof(float);

    float *d_in=NULL;
    float *d_out=NULL;
    hipMalloc((void**)&d_in,bytes_in);
    hipMalloc((void**)&d_out,bytes_out);

    int const  x=32;
    int const  y=16;
    //int const  share_x=((x>>1)+1);
    //int const  share_y=(y>>1)+1;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);

    hipMemcpy(d_in,in_image,bytes_in,hipMemcpyHostToDevice);
    //kernel_doublesizebyshare<<<grid,block,share_x*share_y*3*channels*sizeof(float)>>>(d_out,d_in,ow,oh,weight,height,channels);
    kernel_doublesize<<<grid,block>>>(d_out,d_in,ow,oh,weight,channels);
    hipMemcpy(out_image,d_out,bytes_out,hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

}

void halfsize_by_cuda(float * const out_image,float const  * const in_image, int const weight,int const height,int const channels)
{
    int ow=(weight+1)>>1;
    int oh=(height+1)>>1;
    int const size_in=weight*height;
    int const size_out=ow*oh;
    int const bytes_in=size_in*channels* sizeof(float);
    int const bytes_out=size_out*channels* sizeof(float);

    float *d_in=NULL;
    float *d_out=NULL;
    hipMalloc((void**)&d_in,bytes_in);
    hipMalloc((void**)&d_out,bytes_out);

    int const  x=32;
    int const  y=4;
    int const   share_x=x*4;
    int const   share_y=y*2;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    hipMemcpy(d_in,in_image,bytes_in,hipMemcpyHostToDevice);
    kernel_halfsizebyshare1<<<grid,block,share_x*share_y*channels* sizeof(float)>>>(d_out,d_in,ow,oh,weight,height,channels);
    hipMemcpy(out_image,d_out,bytes_out,hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}

