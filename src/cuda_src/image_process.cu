/**
 * @desc    图像处理函数加速
 * @author  杨丰拓
 * @date    2019-04-16
 * @email   yangfengtuo@163.com
*/
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
void compare(float *const out_image, float const *const out, int const ow, int const oh, int const ic) {
    bool success = 1;
    for (int j = 0; j < oh; ++j) {
        for (int i = 0; i < ow; ++i) {
            for (int k = 0; k < ic; ++k) {
                float a = out[j * ow * ic + i * ic + k];
                float b = out_image[j * ow * ic + i * ic + k];
                if (std::abs(a - b) > 0.000001)
                    //if(a!=b)
                {
                    printf("idx:%d\t", j * ow * ic + i * ic + k);
                    printf("cpu:\t%1.18lf\tgpu:\t%1.18lf\n", a, b);
                    success = 0;
                }
            }
        }
    }
    if (success)std::cout << "gpu加速后的计算结果与cpu计算的结果一致!" << std::endl;
}

void compare_split(float const *const in_image, float *out_1, float *out_2, float *out_3, int const weight,
                   int const height) {
    bool success = 1;
    for (int j = 0; j < height; ++j) {
        for (int i = 0; i < weight; ++i) {
            float a_0 = in_image[j * weight * 3 + i * 3];
            float a_1 = in_image[j * weight * 3 + i * 3 + 1];
            float a_2 = in_image[j * weight * 3 + i * 3 + 2];

            float b_0 = out_1[j * weight + i];
            float b_1 = out_2[j * weight + i];
            float b_2 = out_3[j * weight + i];
            if (a_0 != b_0) {
                printf("idx:%d\t%f\t%f\n", j * weight + i, a_0, b_0);
                success = 0;
            }
            if (!success) {
                std::cout << "第一通道分离失败" << std::endl;
                exit(1);
            }
            if (a_1 != b_1) {
                printf("idx:%d\t%f\t%f\n", j * weight + i, a_1, b_1);
                success = 0;
            }
            if (!success) {
                std::cout << "第二通道分离失败" << std::endl;
                exit(1);
            }
            if (a_2 != b_2) {
                printf("idx:%d\t%f\t%f\n", j * weight + i, a_2, b_2);
                success = 0;
            }
            if (!success) {
                std::cout << "第三通道分离失败" << std::endl;
                exit(1);
            }
        }
    }
    if (success)std::cout << "分离通道成功" << std::endl;
}

void gpuzero(float *a, float *b, float *c, size_t const bytes) {
    hipMemset(a, 0, bytes);
    hipMemset(b, 0, bytes);
    hipMemset(c, 0, bytes);
}

void cpuzero(float *a, float *b, float *c, size_t const bytes) {
    memset(a, 0, bytes);
    memset(b, 0, bytes);
    memset(c, 0, bytes);
}

void gpu2cpu3(float *h_in1, float *d_in1, float *h_in2, float *d_in2, float *h_in3, float *d_in3,
              size_t const bytes_channels) {
    hipMemcpy(h_in1, d_in1, bytes_channels, hipMemcpyDeviceToHost);
    hipMemcpy(h_in2, d_in2, bytes_channels, hipMemcpyDeviceToHost);
    hipMemcpy(h_in3, d_in3, bytes_channels, hipMemcpyDeviceToHost);
}
__global__ void warmup(void)
{}
/*
__global__ void kernel_desaturate_alpha(float *out,float const *in, const int size,const int type)
{
    extern __shared__   float s[];
    int in_idx = threadIdx.x  + blockIdx.x * blockDim.x * 8 ;
    int out_idx = threadIdx.x+ blockIdx.x * blockDim.x * 4 ;
    int tid=threadIdx.x;
    int stride=tid*4;
    int stride1=stride+blockDim.x*4;
    if (in_idx< size * 4)
    {
        s[tid]=in[in_idx];
        s[tid+blockDim.x]=in[in_idx+blockDim.x];
        s[tid+blockDim.x*2]=in[in_idx+blockDim.x*2];
        s[tid+blockDim.x*3]=in[in_idx+blockDim.x*3];
        s[tid+blockDim.x*4]=in[in_idx+blockDim.x*4];
        s[tid+blockDim.x*5]=in[in_idx+blockDim.x*5];
        s[tid+blockDim.x*6]=in[in_idx+blockDim.x*6];
        s[tid+blockDim.x*7]=in[in_idx+blockDim.x*7];
    }
    __syncthreads();

    if(type==0)
    {
        out[out_idx]=max(s[stride+0],max(s[stride+1],s[stride+2]));
        out[out_idx+blockDim.x*2]=max(s[stride1+0],max(s[stride1+1],s[stride1+2]));
    }
    if(type==1)
    {
        float const max_v = max(s[stride+0],max(s[stride+1],s[stride+2]));
        float const min_v = min(s[stride+0],min(s[stride+1],s[stride+2]));
        out[out_idx]=0.5f*(max_v+min_v);
        float const max_s = max(s[stride1+0],max(s[stride1+1],s[stride1+2]));
        float const min_s = min(s[stride1+0],min(s[stride1+1],s[stride1+2]));
        out[out_idx+blockDim.x*2]=0.5f*(max_s+min_s);
    }
    if(type==2)
    {
        out[out_idx]=0.21f * s[stride+0] + 0.72f * s[stride+1] + 0.07f * s[stride+2];
        out[out_idx+blockDim.x*2]=0.21f * s[stride1+0] + 0.72f * s[stride1+1] + 0.07f * s[stride1+2];
    }
    if(type==3)
    {
        out[out_idx]=0.30f * s[stride+0] + 0.59f * s[stride+1] + 0.11f * s[stride+2];
        out[out_idx+blockDim.x*2]=0.30f * s[stride1+0] + 0.59f * s[stride1+1] + 0.11f * s[stride1+2];
    }
    if(type==4)
    {
        out[out_idx]=((float)(s[stride+0] + s[stride+1] + s[stride+2])) / 3.0f;
        out[out_idx+blockDim.x*2]=((float)(s[stride1+0] + s[stride1+1] + s[stride1+2])) / 3.0f;
    }
    out[out_idx+tid+1]=s[stride+3];
    out[out_idx+blockDim.x*2+tid+1]=s[stride1+3];
}
__global__ void kernel_desaturate(float *out,float const *in, const int size,const int type)
{
    extern __shared__   float s[];
    int in_idx = threadIdx.x  + blockIdx.x * blockDim.x * 6 ;
    int out_idx = threadIdx.x+ blockIdx.x * blockDim.x * 2 ;
    int tid=threadIdx.x;
    int stride=tid*3;
    int stride1=stride+blockDim.x*3;

    if (in_idx< size * 3)
    {
        s[tid]=in[in_idx];
        s[tid+blockDim.x]=in[in_idx+blockDim.x];
        s[tid+blockDim.x*2]=in[in_idx+blockDim.x*2];
        s[tid+blockDim.x*3]=in[in_idx+blockDim.x*3];
        s[tid+blockDim.x*4]=in[in_idx+blockDim.x*4];
        s[tid+blockDim.x*5]=in[in_idx+blockDim.x*5];
    }
    __syncthreads();
    if(type==0)
    {
        out[out_idx]=max(s[stride+0],max(s[stride+1],s[stride+2]));
        out[out_idx+blockDim.x]=max(s[stride1+0],max(s[stride1+1],s[stride1+2]));
    }
    if(type==1)
    {
        float const max_v = max(s[stride+0],max(s[stride+1],s[stride+2]));
        float const min_v = min(s[stride+0],min(s[stride+1],s[stride+2]));
        out[out_idx]=0.5f*(max_v+min_v);
        float const max_s = max(s[stride1+0],max(s[stride1+1],s[stride1+2]));
        float const min_s = min(s[stride1+0],min(s[stride1+1],s[stride1+2]));
        out[out_idx+blockDim.x]=0.5f*(max_s+min_s);
    }
    if(type==2)
    {
        out[out_idx]=0.21f * s[stride+0] + 0.72f * s[stride+1] + 0.07f * s[stride+2];
        out[out_idx+blockDim.x]=0.21f * s[stride1+0] + 0.72f * s[stride1+1] + 0.07f * s[stride1+2];
    }
    if(type==3)
    {
        out[out_idx]=0.30f * s[stride+0] + 0.59f * s[stride+1] + 0.11f * s[stride+2];
        out[out_idx+blockDim.x]=0.30f * s[stride1+0] + 0.59f * s[stride1+1] + 0.11f * s[stride1+2];
    }
    if(type==4)
    {
        out[out_idx]=((float)(s[stride+0] + s[stride+1] + s[stride+2])) / 3.0f;
        out[out_idx+blockDim.x]=((float)(s[stride1+0] + s[stride1+1] + s[stride1+2])) / 3.0f;
    }

}


void desaturate_by_cuda(float  * const out_image,float const *in_image,const int pixel_amount, const int type,const bool alpha)
{
    float *d_in=NULL;
    float *d_out=NULL;

    int bytes_in=pixel_amount*(3+alpha)*sizeof(float);
    int bytes_out=pixel_amount*(1+alpha)* sizeof(float);
    const int  blocksize=256;
    dim3 block(blocksize,1,1);
    dim3 grid((pixel_amount-1+blocksize*2)/(blocksize*2),1,1);
    cudaMalloc(&d_in,bytes_in);
    cudaMalloc(&d_out,bytes_out);
    cudaMemcpy(d_in,in_image,bytes_in,cudaMemcpyHostToDevice);
    if(alpha)
    {
        kernel_desaturate_alpha<<<grid,block,blocksize*4* sizeof(float)>>>(d_out,d_in,pixel_amount,type);
    }
    else
    {
        kernel_desaturate<<<grid,block,blocksize*6* sizeof(float)>>>(d_out,d_in,pixel_amount,type);
    }
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);

    cudaFree(d_in);
    cudaFree(d_out);
}
*/


/******************************************************************************************/
///功能：图片放大两倍
/*  函数名                         线程块大小       耗费时间
 *  kernel_doublesize               3.678ms	    [32,4,1]
 *  kernel_doublesize1              3.67ms	    [32,4,1]
 *  kernel_doublesize2              3.532ms	    [32,4,1]**
 *  kernel_doublesizebyshare        5.265ms	    [32,8,1]
 *  kernel_doublesizebyshare1       4.737ms	    [64,8,1]
 *  kernel_doublesizebyshare2       3.98ms	    [32,8,1]
 */
/******************************************************************************************/
__global__ void kernel_doublesize(float *out,float *in,int const image_x,int const image_y,int const iw,int const ic)
{
    int out_x = threadIdx.x + blockIdx.x * blockDim.x * ic;
    int out_y = threadIdx.y + blockIdx.y * blockDim.y;
    for (int c = 0; c <ic ; ++c)
    {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<image_y&&fact_x<image_x*ic)
        {
            int idx=fact_x+out_y*image_x*ic;
            bool nexty=(out_y+1)<image_y;
            bool nextx=(fact_x+ic)<(image_x*ic);
            int yoff[2]={ic*iw*(out_y>>1),
                         ic*iw*((out_y+nexty)>>1)};
            int xoff[2]={((fact_x/ic)>>1)*ic+fact_x%ic,
                         (((fact_x/ic)+nextx)>>1)*ic+fact_x%ic};
            int index[4]={yoff[0]+xoff[0],
                          yoff[0]+xoff[1],
                          yoff[1]+xoff[0],
                          yoff[1]+xoff[1]};
            out[idx]=0.25f*(in[index[0]]+in[index[1]]+in[index[2]]+in[index[3]]);
        }
    }
}
__global__ void kernel_doublesize1(float *out,float *in,int const image_x,int const image_y,int const iw,int const ic)
{
    int out_x = threadIdx.x + blockIdx.x * blockDim.x * ic*2;
    int out_y = threadIdx.y + blockIdx.y * blockDim.y;
    for (int c = 0; c <ic*2 ; ++c)
    {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<image_y&&fact_x<image_x*ic)
        {
            int idx=fact_x+out_y*image_x*ic;
            bool nexty=(out_y+1)<image_y;
            bool nextx=(fact_x+ic)<(image_x*ic);
            int yoff[2]={ic*iw*(out_y>>1),
                         ic*iw*((out_y+nexty)>>1)};
            int xoff[2]={((fact_x/ic)>>1)*ic+fact_x%ic,
                         (((fact_x/ic)+nextx)>>1)*ic+fact_x%ic};
            int index[4]={yoff[0]+xoff[0],
                          yoff[0]+xoff[1],
                          yoff[1]+xoff[0],
                          yoff[1]+xoff[1]};
            out[idx]=0.25f*(in[index[0]]+in[index[1]]+in[index[2]]+in[index[3]]);
        }
    }
}
__global__ void kernel_doublesize2(float *out,float *in,int const image_x,int const image_y,int const iw,int const ic)
{
    int out_x = threadIdx.x + blockIdx.x * blockDim.x * ic*3;
    int out_y = threadIdx.y + blockIdx.y * blockDim.y;
    for (int c = 0; c <ic*3 ; ++c)
    {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<image_y&&fact_x<image_x*ic)
        {
            int idx=fact_x+out_y*image_x*ic;
            bool nexty=(out_y+1)<image_y;
            bool nextx=(fact_x+ic)<(image_x*ic);
            int yoff[2]={ic*iw*(out_y>>1),
                         ic*iw*((out_y+nexty)>>1)};
            int xoff[2]={((fact_x/ic)>>1)*ic+fact_x%ic,
                         (((fact_x/ic)+nextx)>>1)*ic+fact_x%ic};
            int index[4]={yoff[0]+xoff[0],
                          yoff[0]+xoff[1],
                          yoff[1]+xoff[0],
                          yoff[1]+xoff[1]};
            out[idx]=0.25f*(in[index[0]]+in[index[1]]+in[index[2]]+in[index[3]]);
        }
    }
}
__global__ void kernel_doublesize_dim3(float *out,float *in,int const image_x,int const image_y,int const iw)
{
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*2;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int out_z=threadIdx.z;

    if(out_x<image_x&&out_y<image_y)
    {
        int idx=out_y*image_x*blockDim.z+out_x*blockDim.z+out_z;

        const bool nexty=(out_y+1)<image_y;
        const bool nextx=(out_x+1)<image_x;
        int yoff[2]={blockDim.z*iw*(out_y>>1),blockDim.z*iw*((out_y+nexty)>>1)};
        int xoff[2]={blockDim.z*(out_x>>1),blockDim.z*((out_x+nextx)>>1)};
        int index[4]={yoff[0]+xoff[0]+out_z,
                      yoff[0]+xoff[1]+out_z,
                      yoff[1]+xoff[0]+out_z,
                      yoff[1]+xoff[1]+out_z};
        out[idx]=0.25f*(in[index[0]]+in[index[1]]+in[index[2]]+in[index[3]]);

        int idx_2=out_y*image_x*blockDim.z+(out_x+blockDim.x)*blockDim.z+out_z;
        const bool nextx_2=(out_x+blockDim.x+1)<image_x;
        int xoff_2[2]={blockDim.z*((out_x+blockDim.x)>>1),blockDim.z*((out_x+blockDim.x+nextx_2)>>1)};
        int index_2[4]={yoff[0]+xoff_2[0]+out_z,
                      yoff[0]+xoff_2[1]+out_z,
                      yoff[1]+xoff_2[0]+out_z,
                      yoff[1]+xoff_2[1]+out_z};
        out[idx_2]=0.25f*(in[index_2[0]]+in[index_2[1]]+in[index_2[2]]+in[index_2[3]]);


    }

}
__global__ void kernel_doublesizebyshare(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    extern __shared__ float  data[];
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;

    int share_x=(blockDim.x>>1)+1;//共享内存块x维（需乘ic）
    int share_y=(blockDim.y>>1)+1;//共享内存块y维
    int share_fact_x=share_x*ic;
    int share_idx_x;
    int share_idx_y= threadIdx.y;//共享内存块内y维索引
    int in_x0 = ((blockIdx.x * blockDim.x) >> 1) * ic;
    int in_y0 = (blockIdx.y * blockDim.y) >> 1;
    int x,y,c,fact_x;

    for ( c = 0; c <ic ; ++c)
    {
        share_idx_x = threadIdx.x + blockDim.x * c;//共享内存块内x索引
        if (share_idx_x < share_fact_x && share_idx_y < share_y)
        {
            x = min(in_x0 + share_idx_x, iw * ic - ic + share_idx_x % ic);
            y = min(in_y0 + share_idx_y, ih - 1);
            data[share_idx_y * share_fact_x + share_idx_x] = in[y * iw * ic + x];
        }

    }
    __syncthreads();
    for ( c = 0; c <ic ; ++c)
    {
        fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic)
        {
            share_idx_x = threadIdx.x + blockDim.x * c;
            int yoff[2]={(share_idx_y>>1)*share_fact_x,((share_idx_y+1)>>1)*share_fact_x};
            int xoff[2]={(share_idx_x/ic>>1)*ic+share_idx_x%ic,
                         ((share_idx_x/ic+1)>>1)*ic+share_idx_x%ic};
            int out_idx=out_y*ow*ic+fact_x;
            int index[4]={yoff[0]+xoff[0],
                          yoff[0]+xoff[1],
                          yoff[1]+xoff[0],
                          yoff[1]+xoff[1]};
            out[out_idx]=0.25f*(data[index[0]]+data[index[1]]+data[index[2]]+data[index[3]]);
        }
    }
}
__global__ void kernel_doublesizebyshare1(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    extern __shared__ float  data[];
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic*2;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;

    int share_x=(blockDim.x>>1)+1;//共享内存块x维（需乘ic）
    int share_y=(blockDim.y>>1)+1;//共享内存块y维
    int share_fact_x=share_x*ic*2;
    int share_idx_x;
    int share_idx_y= threadIdx.y;//共享内存块内y维索引
    int in_x0 = ((blockIdx.x * blockDim.x*2) >> 1) * ic;
    int in_y0 = (blockIdx.y * blockDim.y) >> 1;
    int x,y,c,fact_x;

    for ( c = 0; c <ic*2 ; ++c)
    {
        share_idx_x = threadIdx.x + blockDim.x * c;//共享内存块内x索引
        if (share_idx_x < share_fact_x && share_idx_y < share_y)
        {
            x = min(in_x0 + share_idx_x, iw * ic - ic + share_idx_x % ic);
            y = min(in_y0 + share_idx_y, ih - 1);
            data[share_idx_y * share_fact_x + share_idx_x] = in[y * iw * ic + x];
        }

    }
    __syncthreads();
    for ( c = 0; c <ic*2 ; ++c)
    {
        fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic)
        {
            share_idx_x = threadIdx.x + blockDim.x * c;
            int yoff[2]={(share_idx_y>>1)*share_fact_x,((share_idx_y+1)>>1)*share_fact_x};
            int xoff[2]={(share_idx_x/ic>>1)*ic+share_idx_x%ic,
                         ((share_idx_x/ic+1)>>1)*ic+share_idx_x%ic};
            int out_idx=out_y*ow*ic+fact_x;
            int index[4]={yoff[0]+xoff[0],
                          yoff[0]+xoff[1],
                          yoff[1]+xoff[0],
                          yoff[1]+xoff[1]};
            out[out_idx]=0.25f*(data[index[0]]+data[index[1]]+data[index[2]]+data[index[3]]);
        }
    }
}
__global__ void kernel_doublesizebyshare2(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    extern __shared__ float  data[];
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic*3;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;

    int share_x=(blockDim.x>>1)+1;//共享内存块x维（需乘ic）
    int share_y=(blockDim.y>>1)+1;//共享内存块y维
    int share_fact_x=share_x*ic*3;
    int share_idx_x;
    int share_idx_y= threadIdx.y;//共享内存块内y维索引
    int in_x0 = ((blockIdx.x * blockDim.x*3) >> 1) * ic;
    int in_y0 = (blockIdx.y * blockDim.y) >> 1;
    int x,y,c,fact_x;

    for ( c = 0; c <ic*3 ; ++c)
    {
        share_idx_x = threadIdx.x + blockDim.x * c;//共享内存块内x索引
        if (share_idx_x < share_fact_x && share_idx_y < share_y)
        {
            x = min(in_x0 + share_idx_x, iw * ic - ic + share_idx_x % ic);
            y = min(in_y0 + share_idx_y, ih - 1);
            data[share_idx_y * share_fact_x + share_idx_x] = in[y * iw * ic + x];
        }

    }
    __syncthreads();
    for ( c = 0; c <ic*3 ; ++c)
    {
        fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic)
        {
            share_idx_x = threadIdx.x + blockDim.x * c;
            int yoff[2]={(share_idx_y>>1)*share_fact_x,((share_idx_y+1)>>1)*share_fact_x};
            int xoff[2]={(share_idx_x/ic>>1)*ic+share_idx_x%ic,
                         ((share_idx_x/ic+1)>>1)*ic+share_idx_x%ic};
            int out_idx=out_y*ow*ic+fact_x;
            int index[4]={yoff[0]+xoff[0],
                          yoff[0]+xoff[1],
                          yoff[1]+xoff[0],
                          yoff[1]+xoff[1]};
            out[out_idx]=0.25f*(data[index[0]]+data[index[1]]+data[index[2]]+data[index[3]]);
        }
    }
}

/******************************************************************************************/
///功能：图片缩小两倍
/*  函数名                            线程块大小       耗费时间
 *kernel_halfsize		            636.275us	    [32,8,1]
 *kernel_halfsize1                  634.383us	    [32,8,1]**
 *kernel_halfsize2                  641.6us	        [32,8,1]
 *kernel_halfsizebyshare	    	643.698us	    [32,4,1]
 *kernel_halfsizebyshare1	  		671.245us	    [32,4,1]
 */
/******************************************************************************************/

__global__ void kernel_halfsize(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int stride=iw*ic;

    for(int c=0;c<ic;c++)
    {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic) {
            int irow1 = out_y * 2 * stride;
            int irow2 = irow1 + stride * (out_y * 2 + 1 < ih);
            int icol1 = (fact_x / ic) * 2 * ic + fact_x % ic;
            int icol2 = min((icol1 + ic), (iw * ic - ic + fact_x % ic));
            int index[4] = {irow1 + icol1,
                            irow1 + icol2,
                            irow2 + icol1,
                            irow2 + icol2};
            int out_idx = out_y * ow*ic + fact_x;
            out[out_idx] = 0.25f * (in[index[0]] + in[index[1]] + in[index[2]] + in[index[3]]);
        }
    }
}
__global__ void kernel_halfsize1(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    //若需要展开ic*3重循环只需修改out_x=threadIdx.x+blockIdx.x*blockDim.x*ic*3;以及for(int c=0;c<ic*3;c++)即可，同时应修改网格大小
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic*2;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int stride=iw*ic;

    for(int c=0;c<ic*2;c++)
    {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic) {
            int irow1 = out_y * 2 * stride;
            int irow2 = irow1 + stride * (out_y * 2 + 1 < ih);
            int icol1 = (fact_x / ic) * 2 * ic + fact_x % ic;
            int icol2 = min((icol1 + ic), (iw * ic - ic + fact_x % ic));
            int index[4] = {irow1 + icol1,
                            irow1 + icol2,
                            irow2 + icol1,
                            irow2 + icol2};
            int out_idx = out_y * ow*ic + fact_x;
            out[out_idx] = 0.25f * (in[index[0]] + in[index[1]] + in[index[2]] + in[index[3]]);
        }
    }
}
__global__ void kernel_halfsize2(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    //若需要展开ic*3重循环只需修改out_x=threadIdx.x+blockIdx.x*blockDim.x*ic*3;以及for(int c=0;c<ic*3;c++)即可，同时应修改网格大小
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic*3;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int stride=iw*ic;

    for(int c=0;c<ic*3;c++)
    {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic) {
            int irow1 = out_y * 2 * stride;
            int irow2 = irow1 + stride * (out_y * 2 + 1 < ih);
            int icol1 = (fact_x / ic) * 2 * ic + fact_x % ic;
            int icol2 = min((icol1 + ic), (iw * ic - ic + fact_x % ic));
            int index[4] = {irow1 + icol1,
                            irow1 + icol2,
                            irow2 + icol1,
                            irow2 + icol2};
            int out_idx = out_y * ow*ic + fact_x;
            out[out_idx] = 0.25f * (in[index[0]] + in[index[1]] + in[index[2]] + in[index[3]]);
        }
    }
}
__global__ void kernel_halfsizebyshare(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    extern __shared__ float data[];
    int block_stride=blockDim.x*ic;//线程块x维间隔
    int out_x=threadIdx.x+blockIdx.x*block_stride;//输出的x维起始索引
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;//输出的y位索引
    int stride=iw*ic;//输入图像的行索引的最大值

    int in_x0=blockIdx.x*block_stride*2;//输入图像x维的起始点
    int in_y0=blockIdx.y*blockDim.y*2;//输入图像y维的起始点
    int in_x1=in_x0+block_stride;
    int in_y1=in_y0+blockDim.y;

    int share_x=blockDim.x*2*ic;//共享块内x维最大像素点个数
    for (int c = 0; c < ic; ++c)
    {

        int fact_x_s=threadIdx.x+blockDim.x*c;
        int channel=fact_x_s%ic;//第几个通道
        int x_s=fact_x_s+block_stride;
        int y_s0=threadIdx.y*share_x;
        int y_s1=y_s0+blockDim.y*share_x;
        int fact_iw=channel+stride-ic;
        int x0=min(in_x0+fact_x_s,fact_iw);
        int x1=min(in_x1+fact_x_s,fact_iw);
        int y0=min(in_y0+threadIdx.y,ih-1)*stride;
        int y1=min(in_y1+threadIdx.y,ih-1)*stride;

        int deta=((fact_x_s/ic)%2)*block_stride;//像素点的x坐标是否为奇数

        int x_fs0=(fact_x_s/ic>>1)*ic+channel+deta;//共享内存内存储第一个x坐标
        int x_fs1=(x_s/ic>>1)*ic+channel+deta;//共享内存内存储第二个x坐标

        data[y_s0+x_fs0]=in[y0+x0];
        data[y_s0+x_fs1]=in[y0+x1];
        data[y_s1+x_fs0]=in[y1+x0];
        data[y_s1+x_fs1]=in[y1+x1];;
    }
    __syncthreads();
    for (int c = 0; c <ic ; ++c)
    {
        int fact_x=out_x+blockDim.x*c;

        if(out_y<oh&&fact_x<ow*ic)
        {
            int srow1=threadIdx.y*2*share_x;
            int srow2=srow1+share_x;

            int scol1=threadIdx.x+blockDim.x*c;
            int scol2=scol1+block_stride;
            int index[4] = {srow1 + scol1,
                            srow1 + scol2,
                            srow2 + scol1,
                            srow2 + scol2};
            int out_idx = out_y * ow*ic + fact_x;
            out[out_idx] = 0.25f * (data[index[0]] + data[index[1]] + data[index[2]] + data[index[3]]);
        }
    }
}

__global__ void kernel_halfsizebyshare1(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic)
{
    extern __shared__ float data[];
    int block_stride=blockDim.x*ic*2;//线程块x维间隔
    int out_x=threadIdx.x+blockIdx.x*block_stride;//输出的x维起始索引
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;//输出的y位索引
    int stride=iw*ic;//输入图像的行索引的最大值

    int in_x0=blockIdx.x*block_stride*2;//输入图像x维的起始点
    int in_y0=blockIdx.y*blockDim.y*2;//输入图像y维的起始点
    int in_x1=in_x0+block_stride;
    int in_y1=in_y0+blockDim.y;

    int share_x=blockDim.x*4*ic;//共享块内x维最大像素点个数
    for (int c = 0; c < ic*2; ++c)
    {

        int fact_x_s=threadIdx.x+blockDim.x*c;
        int channel=fact_x_s%ic;//第几个通道
        int x_s=fact_x_s+block_stride;
        int y_s0=threadIdx.y*share_x;
        int y_s1=y_s0+blockDim.y*share_x;
        int fact_iw=channel+stride-ic;
        int x0=min(in_x0+fact_x_s,fact_iw);
        int x1=min(in_x1+fact_x_s,fact_iw);
        int y0=min(in_y0+threadIdx.y,ih-1)*stride;
        int y1=min(in_y1+threadIdx.y,ih-1)*stride;

        int deta=((fact_x_s/ic)%2)*block_stride;//像素点的x坐标是否为奇数

        int x_fs0=(fact_x_s/ic>>1)*ic+channel+deta;//共享内存内存储第一个x坐标
        int x_fs1=(x_s/ic>>1)*ic+channel+deta;//共享内存内存储第二个x坐标

        data[y_s0+x_fs0]=in[y0+x0];
        data[y_s0+x_fs1]=in[y0+x1];
        data[y_s1+x_fs0]=in[y1+x0];
        data[y_s1+x_fs1]=in[y1+x1];;
    }
    __syncthreads();
    for (int c = 0; c <ic*2 ; ++c)
    {
        int fact_x=out_x+blockDim.x*c;

        if(out_y<oh&&fact_x<ow*ic)
        {
            int srow1=threadIdx.y*2*share_x;
            int srow2=srow1+share_x;

            int scol1=threadIdx.x+blockDim.x*c;
            int scol2=scol1+block_stride;
            int index[4] = {srow1 + scol1,
                            srow1 + scol2,
                            srow2 + scol1,
                            srow2 + scol2};
            int out_idx = out_y * ow*ic + fact_x;
            out[out_idx] = 0.25f * (data[index[0]] + data[index[1]] + data[index[2]] + data[index[3]]);
        }
    }
}

/******************************************************************************************/
///功能：分离颜色通道
/*  函数名                            线程块大小       耗费时间
 *kernel_split		                 [32,4,1]         1.071ms
 *kernel_split1                      [32,4,1]         1.06ms
 *kernel_split2                      [32,4,1]         1.058ms
 *kernel_splitbyshare	    	     [32,8,1]         1.064ms
 *kernel_splitbyshare1	  		     [32,8,1]         1.059ms
 *kernel_splitbyshare2               [32,4,1]         1.057ms
 */
/******************************************************************************************/
/* 调用示例
 * dim3 block1(x, y, 1);
 * dim3 grid1((weight - 1 + x) / x, (height - 1 + y) / y, 1);
 * kernel_splitbyshare <<< grid1, block1, x * y * 3 * sizeof(float) >>> (d_c_0, d_c_1, d_c_2, d_in, weight, height);
 */
__global__ void kernel_splitbyshare(float *out_channels_0,float *out_channels_1,float *out_channels_2,float * in,int const weight,int const height)
{
    extern __shared__ float data[];
    int out_x=threadIdx.x+blockIdx.x*blockDim.x;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int idx=out_y*weight+out_x;
    int fidx=threadIdx.y*blockDim.x*3+threadIdx.x*3;
    int share_x=blockDim.x*3;//共享块x维长度

    int shidx0=threadIdx.y*share_x+blockDim.x*0+threadIdx.x;
    int shidx1=threadIdx.y*share_x+blockDim.x*1+threadIdx.x;
    int shidx2=threadIdx.y*share_x+blockDim.x*2+threadIdx.x;
    int inidx0=out_y*weight*3+blockIdx.x*share_x+blockDim.x*0+threadIdx.x;
    int inidx1=out_y*weight*3+blockIdx.x*share_x+blockDim.x*1+threadIdx.x;
    int inidx2=out_y*weight*3+blockIdx.x*share_x+blockDim.x*2+threadIdx.x;

    if(out_x<weight&&out_y<height)
    {
        data[shidx0]=in[inidx0];
        data[shidx1]=in[inidx1];
        data[shidx2]=in[inidx2];
        __syncthreads();
        out_channels_0[idx]=data[fidx+0];
        out_channels_1[idx]=data[fidx+1];
        out_channels_2[idx]=data[fidx+2];
    }
}

/* 调用示例
 * dim3 block3(x, y, 1);
 * dim3 grid3((weight - 1 + x*2) / (x*2), (height - 1 + y) / y, 1);
 * kernel_splitbyshare1<<<grid3, block3, x * y * 6 * sizeof(float) >>> (d_c_0, d_c_1, d_c_2, d_in, weight, height);
 */
__global__ void kernel_splitbyshare1(float *out_channels_0,float *out_channels_1,float *out_channels_2,float * in,int const weight,int const height)
{
    extern __shared__ float data[];
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*2;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int idx=out_y*weight+out_x;

    int share_x=blockDim.x*6;//共享块x维最大值
    int shsp=threadIdx.y*share_x+threadIdx.x;//共享块内索引起点（start point）
    int insp=out_y*weight*3+blockIdx.x*share_x+threadIdx.x;//输入数组内索引起点;
    int fidx=threadIdx.y*share_x+threadIdx.x*3;
    int inc=blockDim.x*3;//增量

    int shidx0=shsp+blockDim.x*0;
    int shidx1=shsp+blockDim.x*1;
    int shidx2=shsp+blockDim.x*2;
    int shidx3=shsp+blockDim.x*3;
    int shidx4=shsp+blockDim.x*4;
    int shidx5=shsp+blockDim.x*5;

    int inidx0=insp+blockDim.x*0;
    int inidx1=insp+blockDim.x*1;
    int inidx2=insp+blockDim.x*2;
    int inidx3=insp+blockDim.x*3;
    int inidx4=insp+blockDim.x*4;
    int inidx5=insp+blockDim.x*5;

    if(out_x<weight&&out_y<height)
    {
        data[shidx0]=in[inidx0];
        data[shidx1]=in[inidx1];
        data[shidx2]=in[inidx2];
        data[shidx3]=in[inidx3];
        data[shidx4]=in[inidx4];
        data[shidx5]=in[inidx5];
        __syncthreads();
        out_channels_0[idx]=data[fidx+0];
        out_channels_1[idx]=data[fidx+1];
        out_channels_2[idx]=data[fidx+2];
        out_channels_0[idx+blockDim.x]=data[fidx+inc+0];
        out_channels_1[idx+blockDim.x]=data[fidx+inc+1];
        out_channels_2[idx+blockDim.x]=data[fidx+inc+2];
    }
}

/* 调用示例
 * dim3 block4(x, y, 1);
 * dim3 grid4((weight - 1 + x*3) / (x*3), (height - 1 + y) / y, 1);
 * kernel_splitbyshare2<<<grid4, block4, x * y * 9 * sizeof(float) >>> (d_c_0, d_c_1, d_c_2, d_in, weight, height);
 */
__global__ void kernel_splitbyshare2(float *out_channels_0,float *out_channels_1,float *out_channels_2,float * in,int const weight,int const height)
{
    extern __shared__ float data[];
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*3;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int idx=out_y*weight+out_x;

    int share_x=blockDim.x*9;//共享块x维最大值
    int shsp=threadIdx.y*share_x+threadIdx.x;//共享块内索引起点（start point）
    int insp=out_y*weight*3+blockIdx.x*share_x+threadIdx.x;//输入数组内索引起点;
    int fidx=threadIdx.y*share_x+threadIdx.x*3;
    int inc=blockDim.x*3;//增量
    int inc1=blockDim.x*6;//增量

    int shidx0=shsp+blockDim.x*0;
    int shidx1=shsp+blockDim.x*1;
    int shidx2=shsp+blockDim.x*2;
    int shidx3=shsp+blockDim.x*3;
    int shidx4=shsp+blockDim.x*4;
    int shidx5=shsp+blockDim.x*5;
    int shidx6=shsp+blockDim.x*6;
    int shidx7=shsp+blockDim.x*7;
    int shidx8=shsp+blockDim.x*8;

    int inidx0=insp+blockDim.x*0;
    int inidx1=insp+blockDim.x*1;
    int inidx2=insp+blockDim.x*2;
    int inidx3=insp+blockDim.x*3;
    int inidx4=insp+blockDim.x*4;
    int inidx5=insp+blockDim.x*5;
    int inidx6=insp+blockDim.x*6;
    int inidx7=insp+blockDim.x*7;
    int inidx8=insp+blockDim.x*8;

    if(out_x<weight&&out_y<height)
    {
        data[shidx0]=in[inidx0];
        data[shidx1]=in[inidx1];
        data[shidx2]=in[inidx2];
        data[shidx3]=in[inidx3];
        data[shidx4]=in[inidx4];
        data[shidx5]=in[inidx5];
        data[shidx6]=in[inidx6];
        data[shidx7]=in[inidx7];
        data[shidx8]=in[inidx8];
        __syncthreads();
        out_channels_0[idx]=data[fidx+0];
        out_channels_1[idx]=data[fidx+1];
        out_channels_2[idx]=data[fidx+2];
        out_channels_0[idx+blockDim.x]=data[fidx+inc+0];
        out_channels_1[idx+blockDim.x]=data[fidx+inc+1];
        out_channels_2[idx+blockDim.x]=data[fidx+inc+2];
        out_channels_0[idx+blockDim.x*2]=data[fidx+inc1+0];
        out_channels_1[idx+blockDim.x*2]=data[fidx+inc1+1];
        out_channels_2[idx+blockDim.x*2]=data[fidx+inc1+2];

    }
}

/* 调用示例
 * dim3 block2(x, y, 1);
 * dim3 grid2((weight - 1 + x) / x, (height - 1 + y) / y, 1);
 * kernel_split<<< grid2, block2>>> (d_c_0, d_c_1, d_c_2, d_in, weight, height);
 */
__global__ void kernel_split(float *out_channels_0,float *out_channels_1,float *out_channels_2,float * in,int const weight,int const height)
{
    int out_x=threadIdx.x+blockIdx.x*blockDim.x;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int idx=out_y*weight+out_x;
    int inidx=out_y * weight * 3 + out_x * 3;
    if(out_x<weight&&out_y<height) {
        float a=in[inidx+0];
        float b=in[inidx+1];
        float c=in[inidx+2];
        out_channels_0[idx] = a;
        out_channels_1[idx] = b;
        out_channels_2[idx] = c;
    }
}

/* 调用示例
 * dim3 block5(x, y, 1);
 * dim3 grid5((weight - 1 + x*2) / (x*2), (height - 1 + y) / y, 1);
 * kernel_split1<<< grid5, block5>>> (d_c_0, d_c_1, d_c_2, d_in, weight, height);
 */
__global__ void kernel_split1(float *out_channels_0,float *out_channels_1,float *out_channels_2,float * in,int const weight,int const height)
{
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*2;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int idx=out_y*weight+out_x;
    int inidx=out_y * weight * 3 + out_x * 3;
    if(out_x<weight&&out_y<height) {
        float a=in[inidx+0];
        float b=in[inidx+1];
        float c=in[inidx+2];
        out_channels_0[idx] = a;
        out_channels_1[idx] = b;
        out_channels_2[idx] = c;
        a=in[inidx +blockDim.x*3+ 0];
        b=in[inidx +blockDim.x*3+ 1];
        c=in[inidx +blockDim.x*3+ 2];
        out_channels_0[idx+blockDim.x] = a;
        out_channels_1[idx+blockDim.x] = b;
        out_channels_2[idx+blockDim.x] = c;
    }
}

/* 调用示例
 * dim3 block6(x, y, 1);
 * dim3 grid6((weight - 1 + x*3) / (x*3), (height - 1 + y) / y, 1);
 * kernel_split2<<< grid6, block6>>> (d_c_0, d_c_1, d_c_2, d_in, weight, height);
 */
__global__ void kernel_split2(float *out_channels_0,float *out_channels_1,float *out_channels_2,float * in,int const weight,int const height)
{
    int out_x=threadIdx.x+blockIdx.x*blockDim.x*3;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int idx=out_y*weight+out_x;
    int inidx=out_y * weight * 3 + out_x * 3;
    if(out_x<weight&&out_y<height) {
        float a=in[inidx+0];
        float b=in[inidx+1];
        float c=in[inidx+2];
        out_channels_0[idx] = a;
        out_channels_1[idx] = b;
        out_channels_2[idx] = c;
        a=in[inidx +blockDim.x*3+ 0];
        b=in[inidx +blockDim.x*3+ 1];
        c=in[inidx +blockDim.x*3+ 2];
        out_channels_0[idx+blockDim.x] = a;
        out_channels_1[idx+blockDim.x] = b;
        out_channels_2[idx+blockDim.x] = c;
        a=in[inidx +blockDim.x*6+ 0];
        b=in[inidx +blockDim.x*6+ 1];
        c=in[inidx +blockDim.x*6+ 2];
        out_channels_0[idx+blockDim.x*2] = a;
        out_channels_1[idx+blockDim.x*2] = b;
        out_channels_2[idx+blockDim.x*2] = c;
    }
}
/******************************************************************************************/
///调用核函数实现加速功能
/******************************************************************************************/

void warm(void)
{
    warmup<<<1,1>>>();
}

void double_size_by_cuda(float * const out_image,float const  * const in_image,int const weight,int const height,int const channels,float const * const out)
{
    int const ow=weight<<1;
    int const oh=height<<1;
    int const size_in=weight*height;
    int const size_out=ow*oh;
    size_t const bytes_in=size_in*channels* sizeof(float);
    size_t const bytes_out=size_out*channels* sizeof(float);


    float *d_in=NULL;
    float *d_out=NULL;
    hipMalloc((void**)&d_in,bytes_in);
    hipMalloc((void**)&d_out,bytes_out);
    hipMemcpy(d_in,in_image,bytes_in,hipMemcpyHostToDevice);

    int x=32;
    int y=4;
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    hipMalloc((void**)&d_out,bytes_out);
    kernel_doublesize2<<<grid2,block2>>>(d_out,d_in,ow,oh,weight,channels);
    hipMemcpy(out_image,d_out,bytes_out,hipMemcpyDeviceToHost);

/*
    int x=32;
    int y=16;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_doublesize<<<grid,block>>>(d_out,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);

//缩小block
    x=32;
    y=8;
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x)/x,(oh-1+y)/y,1);
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    kernel_doublesize<<<grid1,block1>>>(d_out1,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    x=32;
    y=4;
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x)/x,(oh-1+y)/y,1);
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    kernel_doublesize<<<grid2,block2>>>(d_out2,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);
//缩小block
    x=32;
    y=32;
    dim3 block3 (x,y,1);
    dim3 grid3 ((ow-1+x)/x,(oh-1+y)/y,1);
    float *d_out3=NULL;
    float *out_image3= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out3,bytes_out);
    kernel_doublesize<<<grid3,block3>>>(d_out3,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image3,d_out3,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image3,out,ow,oh,channels);
//放大block
    x=64;
    y=8;
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x)/x,(oh-1+y)/y,1);
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    kernel_doublesize<<<grid4,block4>>>(d_out4,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
*/
/*
    int x=32;
    int y=16;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_doublesize1<<<grid,block>>>(d_out,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);
//缩小block
    x=32;
    y=8;
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    kernel_doublesize1<<<grid1,block1>>>(d_out1,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    x=32;
    y=4;
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    kernel_doublesize1<<<grid2,block2>>>(d_out2,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);
//缩小block
    x=32;
    y=32;
    dim3 block3 (x,y,1);
    dim3 grid3 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    float *d_out3=NULL;
    float *out_image3= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out3,bytes_out);
    kernel_doublesize1<<<grid3,block3>>>(d_out3,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image3,d_out3,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image3,out,ow,oh,channels);
//放大block
    x=64;
    y=8;
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    kernel_doublesize1<<<grid4,block4>>>(d_out4,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
    */
/*
    int x=32;
    int y=16;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    kernel_doublesize2<<<grid,block>>>(d_out,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);
//缩小block
    x=32;
    y=8;
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    kernel_doublesize2<<<grid1,block1>>>(d_out1,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    int x=32;
    int y=4;
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    kernel_doublesize2<<<grid2,block2>>>(d_out2,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);

//缩小block
    x=32;
    y=32;
    dim3 block3 (x,y,1);
    dim3 grid3 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    float *d_out3=NULL;
    float *out_image3= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out3,bytes_out);
    kernel_doublesize2<<<grid3,block3>>>(d_out3,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image3,d_out3,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image3,out,ow,oh,channels);
//放大block
    x=64;
    y=8;
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    kernel_doublesize2<<<grid4,block4>>>(d_out4,d_in,ow,oh,weight,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
*/
/*
    int x=32;
    int y=16;
    int share_x=((x>>1)+1);
    int share_y=(y>>1)+1;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_doublesizebyshare<<<grid,block,share_x*share_y*channels*sizeof(float)>>>(d_out,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);

//缩小block
    x=32;
    y=8;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x)/x,(oh-1+y)/y,1);
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    kernel_doublesizebyshare<<<grid1,block1,share_x*share_y*channels*sizeof(float)>>>(d_out1,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    x=32;
    y=4;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x)/x,(oh-1+y)/y,1);
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    kernel_doublesizebyshare<<<grid2,block2,share_x*share_y*channels*sizeof(float)>>>(d_out2,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);
//缩小block
    x=32;
    y=32;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block3 (x,y,1);
    dim3 grid3 ((ow-1+x)/x,(oh-1+y)/y,1);
    float *d_out3=NULL;
    float *out_image3= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out3,bytes_out);
    kernel_doublesizebyshare<<<grid3,block3,share_x*share_y*channels*sizeof(float)>>>(d_out3,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image3,d_out3,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image3,out,ow,oh,channels);
//放大block
    x=64;
    y=8;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x)/x,(oh-1+y)/y,1);
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    kernel_doublesizebyshare<<<grid4,block4,share_x*share_y*channels*sizeof(float)>>>(d_out4,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
*/
/*
    int x=32;
    int y=16;
    int share_x=((x>>1)+1);
    int share_y=(y>>1)+1;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_doublesizebyshare1<<<grid,block,share_x*share_y*2*channels*sizeof(float)>>>(d_out,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);
//缩小block
    x=32;
    y=8;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    kernel_doublesizebyshare1<<<grid1,block1,share_x*share_y*2*channels*sizeof(float)>>>(d_out1,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    x=32;
    y=4;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    kernel_doublesizebyshare1<<<grid2,block2,share_x*share_y*2*channels*sizeof(float)>>>(d_out2,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);
//缩小block
    x=32;
    y=32;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block3 (x,y,1);
    dim3 grid3 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    float *d_out3=NULL;
    float *out_image3= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out3,bytes_out);
    kernel_doublesizebyshare1<<<grid3,block3,share_x*share_y*2*channels*sizeof(float)>>>(d_out3,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image3,d_out3,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image3,out,ow,oh,channels);
//放大block
    x=64;
    y=8;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    kernel_doublesizebyshare1<<<grid4,block4,share_x*share_y*2*channels*sizeof(float)>>>(d_out4,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
*/
/*
    int x=32;
    int y=16;
    int share_x=((x>>1)+1);
    int share_y=(y>>1)+1;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    kernel_doublesizebyshare2<<<grid,block,share_x*share_y*3*channels*sizeof(float)>>>(d_out,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);
//缩小block
    x=32;
    y=8;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    kernel_doublesizebyshare2<<<grid1,block1,share_x*share_y*3*channels*sizeof(float)>>>(d_out1,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    x=32;
    y=4;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    kernel_doublesizebyshare2<<<grid2,block2,share_x*share_y*3*channels*sizeof(float)>>>(d_out2,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);
//缩小block
    x=32;
    y=32;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block3 (x,y,1);
    dim3 grid3 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    float *d_out3=NULL;
    float *out_image3= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out3,bytes_out);
    kernel_doublesizebyshare2<<<grid3,block3,share_x*share_y*3*channels*sizeof(float)>>>(d_out3,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image3,d_out3,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image3,out,ow,oh,channels);
//放大block
    x=64;
    y=8;
    share_x=((x>>1)+1);
    share_y=(y>>1)+1;
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    kernel_doublesizebyshare2<<<grid4,block4,share_x*share_y*3*channels*sizeof(float)>>>(d_out4,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
    */
//释放分配的内存
    /*cudaFree(d_out1);
    cudaFree(d_out2);
    cudaFree(d_out3);
    cudaFree(d_out4);
    free(out_image1);
    free(out_image2);
    free(out_image3);
    free(out_image4);*/

    hipFree(d_in);
    hipFree(d_out);
}

void halfsize_by_cuda(float * const out_image,float const  * const in_image,int const weight,int const height,int const channels,float const  * const out)
{
    int ow=(weight+1)>>1;
    int oh=(height+1)>>1;
    int const size_in=weight*height;
    int const size_out=ow*oh;
    size_t const bytes_in=size_in*channels* sizeof(float);
    size_t const bytes_out=size_out*channels* sizeof(float);

    float *d_in=NULL;
    float *d_out=NULL;
    hipMalloc((void**)&d_out,bytes_out);
    hipMalloc((void**)&d_in,bytes_in);
    hipMemcpy(d_in,in_image,bytes_in,hipMemcpyHostToDevice);

    int const x=32;
    int const y=8;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_halfsize1<<<grid,block>>>(d_out,d_in,ow,oh,weight,height,channels);
    hipMemcpy(out_image,d_out,bytes_out,hipMemcpyDeviceToHost);
    //compare(out_image,out,ow,oh,channels);//对比运行结果
/*
    int x=32;
    int y=16;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_halfsize<<<grid,block>>>(d_out,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);
//缩小block
    x=32;
    y=8;
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_halfsize<<<grid1,block1>>>(d_out1,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    x=32;
    y=4;
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_halfsize<<<grid2,block2>>>(d_out2,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);
//缩小block
    x=32;
    y=32;
    float *d_out3=NULL;
    float *out_image3= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out3,bytes_out);
    dim3 block3 (x,y,1);
    dim3 grid3 ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_halfsize<<<grid3,block3>>>(d_out3,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image3,d_out3,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image3,out,ow,oh,channels);
//放大block
    x=64;
    y=8;
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_halfsize<<<grid4,block4>>>(d_out4,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
*/
/*
    int x=32;
    int y=16;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_halfsize1<<<grid,block>>>(d_out,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);
//缩小block
    x=32;
    y=8;
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_halfsize1<<<grid1,block1>>>(d_out1,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    x=32;
    y=4;
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_halfsize1<<<grid2,block2>>>(d_out2,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);
//缩小block
    x=32;
    y=32;
    float *d_out3=NULL;
    float *out_image3= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out3,bytes_out);
    dim3 block3 (x,y,1);
    dim3 grid3 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_halfsize1<<<grid3,block3>>>(d_out3,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image3,d_out3,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image3,out,ow,oh,channels);
//放大block
    x=64;
    y=8;
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_halfsize1<<<grid4,block4>>>(d_out4,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
*/
/*
    int x=32;
    int y=16;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    kernel_halfsize2<<<grid,block>>>(d_out,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);
//缩小block
    x=32;
    y=8;
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    kernel_halfsize2<<<grid1,block1>>>(d_out1,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    x=32;
    y=4;
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    kernel_halfsize2<<<grid2,block2>>>(d_out2,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);
//缩小block
    x=32;
    y=32;
    float *d_out3=NULL;
    float *out_image3= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out3,bytes_out);
    dim3 block3 (x,y,1);
    dim3 grid3 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    kernel_halfsize2<<<grid3,block3>>>(d_out3,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image3,d_out3,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image3,out,ow,oh,channels);
//放大block
    x=64;
    y=8;
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x*3)/(x*3),(oh-1+y)/y,1);
    kernel_halfsize2<<<grid4,block4>>>(d_out4,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
*/
/*
    int x=32;
    int y=16;
    int share_x=x*2;
    int  share_y=y*2;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_halfsizebyshare<<<grid,block,share_x*share_y*channels* sizeof(float)>>>(d_out,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);
//缩小block
    x=32;
    y=8;
    share_x=x*2;
    share_y=y*2;
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_halfsizebyshare<<<grid1,block1,share_x*share_y*channels* sizeof(float)>>>(d_out1,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    x=32;
    y=4;
    share_x=x*2;
    share_y=y*2;
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_halfsizebyshare<<<grid2,block2,share_x*share_y*channels* sizeof(float)>>>(d_out2,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);
//缩小block
    x=32;
    y=32;
    share_x=x*2;
    share_y=y*2;
    float *d_out3=NULL;
    float *out_image3= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out3,bytes_out);
    dim3 block3 (x,y,1);
    dim3 grid3 ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_halfsizebyshare<<<grid3,block3,share_x*share_y*channels* sizeof(float)>>>(d_out3,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image3,d_out3,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image3,out,ow,oh,channels);
//放大block
    x=64;
    y=8;
    share_x=x*2;
    share_y=y*2;
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x)/x,(oh-1+y)/y,1);
    kernel_halfsizebyshare<<<grid4,block4,share_x*share_y*channels* sizeof(float)>>>(d_out4,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
*/
/*
    int x=32;
    int y=16;
    int share_x=x*4;
    int  share_y=y*2;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_halfsizebyshare1<<<grid,block,share_x*share_y*channels* sizeof(float)>>>(d_out,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);
//缩小block
    x=32;
    y=8;
    share_x=x*4;
    share_y=y*2;
    float *d_out1=NULL;
    float *out_image1= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out1,bytes_out);
    dim3 block1 (x,y,1);
    dim3 grid1 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_halfsizebyshare1<<<grid1,block1,share_x*share_y*channels* sizeof(float)>>>(d_out1,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image1,d_out1,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image1,out,ow,oh,channels);
//缩小block
    x=32;
    y=4;
    share_x=x*4;
    share_y=y*2;
    float *d_out2=NULL;
    float *out_image2= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out2,bytes_out);
    dim3 block2 (x,y,1);
    dim3 grid2 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_halfsizebyshare1<<<grid2,block2,share_x*share_y*channels* sizeof(float)>>>(d_out2,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image2,d_out2,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image2,out,ow,oh,channels);
//缩小block
    //x=32;y=32;share_x=x*4;share_y=y*2;无法正常运行

//放大block
    x=64;
    y=8;
    share_x=x*4;
    share_y=y*2;
    float *d_out4=NULL;
    float *out_image4= (float *) malloc(bytes_out);
    cudaMalloc((void**)&d_out4,bytes_out);
    dim3 block4 (x,y,1);
    dim3 grid4 ((ow-1+x*2)/(x*2),(oh-1+y)/y,1);
    kernel_halfsizebyshare1<<<grid4,block4,share_x*share_y*channels* sizeof(float)>>>(d_out4,d_in,ow,oh,weight,height,channels);
    cudaMemcpy(out_image4,d_out4,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image4,out,ow,oh,channels);
*/

    /*cudaFree(d_out1);
    free(out_image1);
    cudaFree(d_out2);
    free(out_image2);
    cudaFree(d_out3);
    free(out_image3);
    cudaFree(d_out4);
    free(out_image4);*/

    hipFree(d_in);
    hipFree(d_out);
}

__global__ void kernel_halfsize_guass(float *out,float *in,int const ow,int const oh,int const iw,int const ih,int const ic,float const *w)
{

    int out_x=threadIdx.x+blockIdx.x*blockDim.x*ic;
    int out_y=threadIdx.y+blockIdx.y*blockDim.y;
    int istride=iw*ic;

    for (int c = 0; c <ic ; ++c)
    {
        int fact_x=out_x+blockDim.x*c;
        if(out_y<oh&&fact_x<ow*ic)
        {
            int out_idx = out_y * ow * ic + fact_x;
            int channels = fact_x % ic;//颜色通道
            int out_xf = fact_x / ic;//输出像素点x坐标
            int ix = out_xf << 1;
            int iy = out_y << 1;
            int row[4], col[4];
            row[0] = max(0, iy - 1) * istride;
            row[1] = iy * istride;
            row[2] = min(iy + 1, (int)ih - 1) * istride;
            row[3] = min(iy + 2, (int)ih - 2) * istride;

            col[0] = max(0, ix - 1) * ic + channels;
            col[1] = ix * ic + channels;
            col[2] = min(ix + 1, (int)iw - 1) * ic + channels;
            col[3] = min(ix + 2, (int)iw - 1) * ic + channels;

            float sum = 0.0f;
            int t=4;
            sum += in[row[0] + col[0]] * w[2];
            sum += in[row[0] + col[1]] * w[1];
            sum += in[row[0] + col[2]] * w[1];
            sum += in[row[0] + col[3]] * w[2];
            //if(out_idx==t)printf("gpu:%1.18f\n",sum);

            sum += in[row[1] + col[0]] * w[1];
            sum += in[row[1] + col[1]] * w[0];
            sum += in[row[1] + col[2]] * w[0];
            sum += in[row[1] + col[3]] * w[1];
            if(out_idx==t)printf("gpu:%1.18f\n",sum);

            /**/
            sum += in[row[2] + col[0]] * w[1];
            if(out_idx==t)printf("gpu:%1.18f\n",sum);
            sum += in[row[2] + col[1]] * w[0];
            if(out_idx==t)printf("gpu:%1.18f\n",sum);
            if(out_idx==t)
            {
                printf("高斯核索引:%d\t",row[2] + col[2]);
                printf("in:%1.18f\t",in[row[2] + col[2]]);
                printf("w:%1.18f\t",w[0]);
                printf("in*w:%1.18f\n",in[row[2] + col[2]] *w[0]);
            }
            sum += in[row[2] + col[2]] * w[0];
            if(out_idx==t)printf("gpu:%1.18f\n",sum);
            sum += in[row[2] + col[3]] * w[1];
            //if(out_idx==t)printf("gpu:%1.18f\n",sum);

            sum += in[row[3] + col[0]] * w[2];
            sum += in[row[3] + col[1]] * w[1];
            sum += in[row[3] + col[2]] * w[1];
            sum += in[row[3] + col[3]] * w[2];
            //if(out_idx==t)printf("gpu:%1.18f\n",sum);

            out[out_idx] = sum / (float)(4 * w[2] + 8 * w[1] + 4 * w[0]);

        }

    }
}


void halfsize_guassian_by_cuda(float * const out_image,float const  * const in_image, int const weight,int const height,int const channels,float sigma2,float const  * const out) {
/*
    int ow=(weight+1)>>1;
    int oh=(height+1)>>1;
    int const size_in=weight*height;
    int const size_out=ow*oh;
    int const bytes_in=size_in*channels* sizeof(float);
    int const bytes_out=size_out*channels* sizeof(float);

    float *d_in=NULL;
    float *d_out=NULL;
    float h_w[3];
    float *d_w=NULL;

    float3 *d_in1=NULL;
    cudaMalloc((void**)&d_in1,bytes_in);
    cudaMemcpy(d_in1,in_image,bytes_in,cudaMemcpyHostToDevice);

    h_w[0] = std::exp(-0.5 / (2.0f * sigma2));
    h_w[1] = std::exp(-2.5f / (2.0 * sigma2));
    h_w[2] = std::exp(-4.5f / (2.0f * sigma2));

    cudaMalloc((void**)&d_in,bytes_in);
    cudaMalloc((void**)&d_out,bytes_out);
    cudaMalloc((void**)&d_w,3* sizeof(float));
    cudaMemcpy(d_w,h_w,3* sizeof(float),cudaMemcpyHostToDevice);
    cudaMemcpy(d_in,in_image,bytes_in,cudaMemcpyHostToDevice);
    int const x=32;
    int const y=16;
    dim3 block (x,y,1);
    dim3 grid ((ow-1+x)/(x),(oh-1+y)/y,1);
    kernel_halfsize_guass<<<grid,block>>>(d_out,d_in,ow,oh,weight,height,channels,d_w);
    //compare(d_out,out,ow,oh,channels);
    float *d_channels_0=NULL;
    float *d_channels_1=NULL;
    float *d_channels_2=NULL;
    size_t bytes_channels=weight*height*sizeof(float);
    cudaMalloc((void**)&d_channels_0,bytes_channels);
    cudaMalloc((void**)&d_channels_1,bytes_channels);
    cudaMalloc((void**)&d_channels_2,bytes_channels);

    //split<<<grid,block,x*y*6*sizeof(float)>>>(d_channels_0,d_channels_1,d_channels_2,d_in,weight,height);
    //sp<<<grid,block>>>(d_in1,d_in);
    cudaFree(d_channels_0);
    cudaFree(d_channels_1);
    cudaFree(d_channels_2);


    cudaMemcpy(out_image,d_out,bytes_out,cudaMemcpyDeviceToHost);
    compare(out_image,out,ow,oh,channels);
    cudaFree(d_in);
    cudaFree(d_out);*/
    int ow = (weight + 1) >> 1;
    int oh = (height + 1) >> 1;
    int const size_in = weight * height;
    int const size_out = ow * oh;
    size_t const bytes_in = size_in * channels * sizeof(float);
    size_t const bytes_out = size_out * channels * sizeof(float);
    size_t const bytes_channels = size_in * sizeof(float);

    float *d_in = NULL;

    float *d_c_0 = NULL;
    float *d_c_1 = NULL;
    float *d_c_2 = NULL;

    hipMalloc((void **) &d_in, bytes_in);
    hipMalloc((void **) &d_c_0, bytes_channels);
    hipMalloc((void **) &d_c_1, bytes_channels);
    hipMalloc((void **) &d_c_2, bytes_channels);

    float *out_1, *out_2, *out_3;
    out_1 = (float *) malloc(bytes_channels);
    out_2 = (float *) malloc(bytes_channels);
    out_3 = (float *) malloc(bytes_channels);

    hipMemcpy(d_in, in_image, bytes_in, hipMemcpyHostToDevice);

    int x;
    int y;
    /*dim3 block0(32, 4, 1);
    dim3 grid0((weight +95) / 96, (height +3) / 4, 1);
    kernel_splitbyshare2<<<grid0, block0, 1152 * sizeof(float) >>> (d_c_0, d_c_1, d_c_2, d_in, weight, height);*/

    float *d_out=NULL;
    float h_w[3];
    float *d_w=NULL;

    h_w[0] = std::exp(-0.5 / (2.0f * sigma2));
    h_w[1] = std::exp(-2.5f / (2.0 * sigma2));
    h_w[2] = std::exp(-4.5f / (2.0f * sigma2));

    hipMalloc((void**)&d_out,bytes_out);
    hipMalloc((void**)&d_w,3* sizeof(float));
    hipMemcpy(d_w,h_w,3* sizeof(float),hipMemcpyHostToDevice);
    int const x1=32;
    int const y1=16;
    dim3 block (x1,y1,1);
    dim3 grid ((ow-1+x1)/(x1),(oh-1+y1)/y1,1);
    kernel_halfsize_guass<<<grid,block>>>(d_out,d_in,ow,oh,weight,height,channels,d_w);
    hipMemcpy(out_image,d_out,bytes_out,hipMemcpyDeviceToHost);
    compare(out_image,out,weight,height,channels);

    free(out_1);
    free(out_2);
    free(out_3);
    hipFree(d_in);
    hipFree(d_c_0);
    hipFree(d_c_1);
    hipFree(d_c_2);

}

  //
